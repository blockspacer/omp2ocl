#include "hip/hip_runtime.h"
/*
--------------------------------------------------------------------
  
  NAS Parallel Benchmarks 2.3 OpenMP C versions - EP

  This benchmark is an OpenMP C version of the NPB EP code.
  
  The OpenMP C versions are developed by RWCP and derived from the serial
  Fortran versions in "NPB 2.3-serial" developed by NAS.

  Permission to use, copy, distribute and modify this software for any
  purpose with or without fee is hereby granted.
  This software is provided "as is" without express or implied warranty.
  
  Send comments on the OpenMP C versions to pdp-openmp@rwcp.or.jp

  Information on OpenMP activities at RWCP is available at:

           http:pdplab.trc.rwcp.or.jppdperf/Omni/
  
  Information on NAS Parallel Benchmarks 2.3 is available at:
  
           http:www.nas.nasa.gov/NAS/NPB/

--------------------------------------------------------------------
*/
/*
--------------------------------------------------------------------

  Author: P. O. Frederickson 
          D. H. Bailey
          A. C. Woo

  OpenMP C version: S. Satoh
  
--------------------------------------------------------------------
*/
#include "npb-C.h"
#include "npbparams.h"

#ifndef __O2G_HEADER__ 

#define __O2G_HEADER__ 

/******************************************/
/* Added codes for OpenMP2GPU translation */
/******************************************/
#include <cutil.h>
#include <math.h>
#define MAX(a,b) (((a) > (b)) ? (a) : (b))

static int gpuNumThreads = BLOCK_SIZE;
static int gpuNumBlocks;
static int gpuNumBlocks1;
static int gpuNumBlocks2;
static int totalNumThreads;
unsigned int gpuGmemSize = 0;
unsigned int gpuSmemSize = 0;
static unsigned int gpuBytes = 0;

#endif 
/* End of __O2G_HEADER__ */



float * gpu__an__main;
int * gpu__k_offset__main;
int * gpu__l__main;
int * gpu__np__main;
/* parameters */
/* global variables */
/* commonstorage */
/* static float qq[NQ];		private copy of q[0:NQ-1] */
static float x[(2*(1<<1))];
#pragma omp threadprivate(x)
static float q[10];
/*  */
/*          E  L  A  P  S  E  D  _  T  I  M  E */
/*  */
float elapsed_time(void )
{
float t;
wtime(( & t));
return t;
}

float start[64];
float elapsed[64];
/*  */
/*             T  I  M  E  R  _  C  L  E  A  R */
/*  */
void timer_clear(int n)
{
elapsed[n]=0.0;
return ;
}

/*  */
/*             T  I  M  E  R  _  S  T  A  R  T */
/*  */
void timer_start(int n)
{
start[n]=elapsed_time();
return ;
}

/*  */
/*             T  I  M  E  R  _  S  T  O  P */
/*  */
void timer_stop(int n)
{
float t;
float now;
now=elapsed_time();
t=(now-start[n]);
elapsed[n]+=t;
return ;
}

/*  */
/*             T  I  M  E  R  _  R  E  A  D */
/*  */
float timer_read(int n)
{
float _ret_val_0;
_ret_val_0=elapsed[n];
return _ret_val_0;
}

static void c_print_results(char * name, char ccclass, int n1, int n2, int n3, int niter, int nthreads, float t, float mops, char * optype, int passed_verification, char * npbversion, char * compiletime, char * cc, char * clink, char * c_lib, char * c_inc, char * cflags, char * clinkflags, char * rand)
{
printf("\n\n %s Benchmark Completed\n", name);
printf(" Class           =                        %c\n", ccclass);
/* as in IS */
if (((n2==0)&&(n3==0)))
{
printf(" Size            =             %12d\n", n1);
}
else
{
printf(" Size            =              %3dx%3dx%3d\n", n1, n2, n3);
}
printf(" Iterations      =             %12d\n", niter);
printf(" Threads         =             %12d\n", nthreads);
printf(" Time in seconds =             %12.2f\n", t);
printf(" Mop/s total     =             %12.2f\n", mops);
printf(" Operation type  = %24s\n", optype);
if (passed_verification)
{
printf(" Verification    =               SUCCESSFUL\n");
}
else
{
printf(" Verification    =             UNSUCCESSFUL\n");
}
printf(" Version         =             %12s\n", npbversion);
printf(" Compile date    =             %12s\n", compiletime);
printf("\n Compile options:\n");
printf("    CC           = %s\n", cc);
printf("    CLINK        = %s\n", clink);
printf("    C_LIB        = %s\n", c_lib);
printf("    C_INC        = %s\n", c_inc);
printf("    CFLAGS       = %s\n", cflags);
printf("    CLINKFLAGS   = %s\n", clinkflags);
printf("    RAND         = %s\n", rand);
/*

	   printf( "\n\n" );
	   printf( " Please send the results of this run to:\n\n" );
	   printf( " NPB Development Team\n" );
	   printf( " Internet: npb@nas.nasa.gov\n \n" );
	   printf( " If email is not available, send this to:\n\n" );
	   printf( " MS T27A-1\n" );
	   printf( " NASA Ames Research Center\n" );
	   printf( " Moffett Field, CA  94035-1000\n\n" );
	   printf( " Fax: 415-604-3957\n\n" );
	
*/
return ;
}

/*
c---------------------------------------------------------------------
c---------------------------------------------------------------------
*/
__device__ static float dev_randlc(float * x, float a)
{
/*
c---------------------------------------------------------------------
c---------------------------------------------------------------------
*/
/*
c---------------------------------------------------------------------
c
c   This routine returns a uniform pseudorandom float precision number in the
c   range (0, 1) by using the linear congruential generator
c
c   x_{k+1} = a x_k  (mod 2^46)
c
c   where 0 < x_k < 2^46 and 0 < a < 2^46.  This scheme generates 2^44 numbers
c   before repeating.  The argument A is the same as 'a' in the above formula,
c   and X is the same as x_0.  A and X must be odd float precision integers
c   in the range (1, 2^46).  The returned value RANDLC is normalized to be
c   between 0 and 1, i.e. RANDLC = 2^(-46) x_1.  X is updated to contain
c   the new seed x_1, so that subsequent calls to RANDLC using the same
c   arguments will generate a continuous sequence.
c
c   This routine should produce the same results on any computer with at least
c   48 mantissa bits in float precision floating point data.  On 64 bit
c   systems, float precision should be disabled.
c
c   David H. Bailey     October 26, 1990
c
c---------------------------------------------------------------------
*/
float t1;
float t2;
float t3;
float t4;
float a1;
float a2;
float x1;
float x2;
float z;
/*
c---------------------------------------------------------------------
c   Break A into two parts such that A = 2^23 A1 + A2.
c---------------------------------------------------------------------
*/
float _ret_val_0;
t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*a);
a1=((int)t1);
a2=(a-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*a1));
/*
c---------------------------------------------------------------------
c   Break X into two parts such that X = 2^23 X1 + X2, compute
c   Z = A1 * X2 + A2 * X1  (mod 2^23), and then
c   X = 2^23 * Z + A2 * X2  (mod 2^46).
c---------------------------------------------------------------------
*/
t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*( * x));
x1=((int)t1);
x2=(( * x)-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*x1));
t1=((a1*x2)+(a2*x1));
t2=((int)(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*t1));
z=(t1-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*t2));
t3=((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*z)+(a2*x2));
t4=((int)((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*t3));
( * x)=(t3-((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0))*t4));
_ret_val_0=((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*( * x));
return _ret_val_0;
}

float randlc(float * x, float a)
{
/*
c---------------------------------------------------------------------
c---------------------------------------------------------------------
*/
/*
c---------------------------------------------------------------------
c
c   This routine returns a uniform pseudorandom float precision number in the
c   range (0, 1) by using the linear congruential generator
c
c   x_{k+1} = a x_k  (mod 2^46)
c
c   where 0 < x_k < 2^46 and 0 < a < 2^46.  This scheme generates 2^44 numbers
c   before repeating.  The argument A is the same as 'a' in the above formula,
c   and X is the same as x_0.  A and X must be odd float precision integers
c   in the range (1, 2^46).  The returned value RANDLC is normalized to be
c   between 0 and 1, i.e. RANDLC = 2^(-46) x_1.  X is updated to contain
c   the new seed x_1, so that subsequent calls to RANDLC using the same
c   arguments will generate a continuous sequence.
c
c   This routine should produce the same results on any computer with at least
c   48 mantissa bits in float precision floating point data.  On 64 bit
c   systems, float precision should be disabled.
c
c   David H. Bailey     October 26, 1990
c
c---------------------------------------------------------------------
*/
float t1;
float t2;
float t3;
float t4;
float a1;
float a2;
float x1;
float x2;
float z;
/*
c---------------------------------------------------------------------
c   Break A into two parts such that A = 2^23 A1 + A2.
c---------------------------------------------------------------------
*/
float _ret_val_0;
t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*a);
a1=((int)t1);
a2=(a-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*a1));
/*
c---------------------------------------------------------------------
c   Break X into two parts such that X = 2^23 X1 + X2, compute
c   Z = A1 * X2 + A2 * X1  (mod 2^23), and then
c   X = 2^23 * Z + A2 * X2  (mod 2^46).
c---------------------------------------------------------------------
*/
t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*( * x));
x1=((int)t1);
x2=(( * x)-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*x1));
t1=((a1*x2)+(a2*x1));
t2=((int)(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*t1));
z=(t1-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*t2));
t3=((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*z)+(a2*x2));
t4=((int)((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*t3));
( * x)=(t3-((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0))*t4));
_ret_val_0=((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*( * x));
return _ret_val_0;
}

/*
c---------------------------------------------------------------------
c---------------------------------------------------------------------
*/
__device__ static void dev_vranlc_n(int n, float * x_seed, float a, float * y, size_t pitch__x)
{
/*
c---------------------------------------------------------------------
c---------------------------------------------------------------------
*/
/*
c---------------------------------------------------------------------
c
c   This routine generates N uniform pseudorandom float precision numbers in
c   the range (0, 1) by using the linear congruential generator
c
c   x_{k+1} = a x_k  (mod 2^46)
c
c   where 0 < x_k < 2^46 and 0 < a < 2^46.  This scheme generates 2^44 numbers
c   before repeating.  The argument A is the same as 'a' in the above formula,
c   and X is the same as x_0.  A and X must be odd float precision integers
c   in the range (1, 2^46).  The N results are placed in Y and are normalized
c   to be between 0 and 1.  X is updated to contain the new seed, so that
c   subsequent calls to VRANLC using the same arguments will generate a
c   continuous sequence.  If N is zero, only initialization is performed, and
c   the variables X, A and Y are ignored.
c
c   This routine is the standard version designed for scalar or RISC systems.
c   However, it should produce the same results on any single processor
c   computer with at least 48 mantissa bits in float precision floating point
c   data.  On 64 bit systems, float precision should be disabled.
c
c---------------------------------------------------------------------
*/
int i;
float x;
float t1;
float t2;
float t3;
float t4;
float a1;
float a2;
float x1;
float x2;
float z;
/*
c---------------------------------------------------------------------
c   Break A into two parts such that A = 2^23 A1 + A2.
c---------------------------------------------------------------------
*/
t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*a);
a1=((int)t1);
a2=(a-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*a1));
x=( * x_seed);
/*
c---------------------------------------------------------------------
c   Generate N results.   This loop is not vectorizable.
c---------------------------------------------------------------------
*/
#pragma loop name vranlc_n#0 
for (i=0; i<n; i ++ )
{
/*
c---------------------------------------------------------------------
c   Break X into two parts such that X = 2^23 X1 + X2, compute
c   Z = A1 * X2 + A2 * X1  (mod 2^23), and then
c   X = 2^23 * Z + A2 * X2  (mod 2^46).
c---------------------------------------------------------------------
*/
t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*x);
x1=((int)t1);
x2=(x-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*x1));
t1=((a1*x2)+(a2*x1));
t2=((int)(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*t1));
z=(t1-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*t2));
t3=((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*z)+(a2*x2));
t4=((int)((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*t3));
x=(t3-((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0))*t4));
( * ((float * )(((char * )y)+(i*pitch__x))))=((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*x);
}
( * x_seed)=x;
return ;
}

void vranlc(int n, float * x_seed, float a, float y[])
{
/*
c---------------------------------------------------------------------
c---------------------------------------------------------------------
*/
/*
c---------------------------------------------------------------------
c
c   This routine generates N uniform pseudorandom float precision numbers in
c   the range (0, 1) by using the linear congruential generator
c
c   x_{k+1} = a x_k  (mod 2^46)
c
c   where 0 < x_k < 2^46 and 0 < a < 2^46.  This scheme generates 2^44 numbers
c   before repeating.  The argument A is the same as 'a' in the above formula,
c   and X is the same as x_0.  A and X must be odd float precision integers
c   in the range (1, 2^46).  The N results are placed in Y and are normalized
c   to be between 0 and 1.  X is updated to contain the new seed, so that
c   subsequent calls to VRANLC using the same arguments will generate a
c   continuous sequence.  If N is zero, only initialization is performed, and
c   the variables X, A and Y are ignored.
c
c   This routine is the standard version designed for scalar or RISC systems.
c   However, it should produce the same results on any single processor
c   computer with at least 48 mantissa bits in float precision floating point
c   data.  On 64 bit systems, float precision should be disabled.
c
c---------------------------------------------------------------------
*/
int i;
float x;
float t1;
float t2;
float t3;
float t4;
float a1;
float a2;
float x1;
float x2;
float z;
/*
c---------------------------------------------------------------------
c   Break A into two parts such that A = 2^23 A1 + A2.
c---------------------------------------------------------------------
*/
t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*a);
a1=((int)t1);
a2=(a-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*a1));
x=( * x_seed);
/*
c---------------------------------------------------------------------
c   Generate N results.   This loop is not vectorizable.
c---------------------------------------------------------------------
*/
#pragma loop name vranlc#0 
for (i=1; i<=n; i ++ )
{
/*
c---------------------------------------------------------------------
c   Break X into two parts such that X = 2^23 X1 + X2, compute
c   Z = A1 * X2 + A2 * X1  (mod 2^23), and then
c   X = 2^23 * Z + A2 * X2  (mod 2^46).
c---------------------------------------------------------------------
*/
t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*x);
x1=((int)t1);
x2=(x-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*x1));
t1=((a1*x2)+(a2*x1));
t2=((int)(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*t1));
z=(t1-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*t2));
t3=((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*z)+(a2*x2));
t4=((int)((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*t3));
x=(t3-((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0))*t4));
y[i]=((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*x);
}
( * x_seed)=x;
return ;
}

/*
--------------------------------------------------------------------
      program EMBAR
c-------------------------------------------------------------------
*/
/*

c   This is the serial version of the APP Benchmark 1,
c   the "embarassingly parallel" benchmark.
c
c   M is the Log_2 of the number of complex pairs of uniform (0, 1) random
c   numbers.  MK is the Log_2 of the size of each batch of uniform random
c   numbers.  MK can be set for convenience on a given system, since it does
c   not affect the results.

*/
__global__ void main_kernel0(float * red__sx, float * red__sy, float * an, int * k_offset, int * l, int * np, float * x, size_t pitch__x)
{
__shared__ float sh__sx[BLOCK_SIZE];
__shared__ float sh__sy[BLOCK_SIZE];
float * x_0;
int i;
int ik;
int k;
int kk;
float t1;
float t2;
float t3;
float t4;
float x1;
float x2;
int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
int _gtid = (threadIdx.x+(_bid*blockDim.x));
x_0=(((float * )x)+_gtid);
sh__sy[threadIdx.x]=0.0F;
sh__sx[threadIdx.x]=0.0F;
k=(_gtid+1);
#pragma omp for nowait
if (k<=( * np))
{
kk=(( * k_offset)+k);
t1=2.71828183E8;
t2=( * an);
/*      Find starting seed t1 for this kk. */
#pragma loop name main#4#0 
for (i=1; i<=100; i ++ )
{
ik=(kk/2);
if (((2*ik)!=kk))
{
t3=dev_randlc(( & t1), t2);
}
if ((ik==0))
{
break;
}
t3=dev_randlc(( & t2), t2);
kk=ik;
}
/*      Compute uniform pseudorandom numbers. */
dev_vranlc_n((2*(1<<1)), ( & t1), 1.220703125E9, x_0, pitch__x);
/* if (TIMERS_ENABLED == TRUE) timer_stop(3); */
/*

c       Compute Gaussian deviates by acceptance-rejection method and 
c       tally counts in concentric square annuli.  This loop is not 
c       vectorizable.

*/
/* if (TIMERS_ENABLED == TRUE) timer_start(2); */
#pragma loop name main#4#1 
for (i=0; i<(1<<1); i ++ )
{
x1=((2.0*( * ((float * )(((char * )x_0)+((2*i)*pitch__x)))))-1.0);
x2=((2.0*( * ((float * )(((char * )x_0)+(((2*i)+1)*pitch__x)))))-1.0);
t1=((x1*x1)+(x2*x2));
if ((t1<=1.0))
{
t2=sqrt(((( - 2.0)*log(t1))/t1));
t3=(x1*t2);
/* Xi */
t4=(x2*t2);
/* Yi */
( * l)=((fabs(t3)>fabs(t4)) ? fabs(t3) : fabs(t4));
/* qq[l] += 1.0;				counts */
sh__sx[threadIdx.x]=(sh__sx[threadIdx.x]+t3);
/* sum of Xi */
sh__sy[threadIdx.x]=(sh__sy[threadIdx.x]+t4);
/* sum of Yi */
}
}
/* if (TIMERS_ENABLED == TRUE) timer_stop(2); */
}
__syncthreads();
if ((threadIdx.x<256))
{
sh__sx[threadIdx.x]+=sh__sx[(threadIdx.x+256)];
sh__sy[threadIdx.x]+=sh__sy[(threadIdx.x+256)];
}
__syncthreads();
if ((threadIdx.x<128))
{
sh__sx[threadIdx.x]+=sh__sx[(threadIdx.x+128)];
sh__sy[threadIdx.x]+=sh__sy[(threadIdx.x+128)];
}
__syncthreads();
if ((threadIdx.x<64))
{
sh__sx[threadIdx.x]+=sh__sx[(threadIdx.x+64)];
sh__sy[threadIdx.x]+=sh__sy[(threadIdx.x+64)];
}
__syncthreads();
if ((threadIdx.x<32))
{
sh__sx[threadIdx.x]+=sh__sx[(threadIdx.x+32)];
sh__sy[threadIdx.x]+=sh__sy[(threadIdx.x+32)];
}
if ((threadIdx.x<16))
{
sh__sx[threadIdx.x]+=sh__sx[(threadIdx.x+16)];
sh__sy[threadIdx.x]+=sh__sy[(threadIdx.x+16)];
}
if ((threadIdx.x<8))
{
sh__sx[threadIdx.x]+=sh__sx[(threadIdx.x+8)];
sh__sy[threadIdx.x]+=sh__sy[(threadIdx.x+8)];
}
if ((threadIdx.x<4))
{
sh__sx[threadIdx.x]+=sh__sx[(threadIdx.x+4)];
sh__sy[threadIdx.x]+=sh__sy[(threadIdx.x+4)];
}
if ((threadIdx.x<2))
{
sh__sx[threadIdx.x]+=sh__sx[(threadIdx.x+2)];
sh__sy[threadIdx.x]+=sh__sy[(threadIdx.x+2)];
}
if ((threadIdx.x<1))
{
sh__sx[threadIdx.x]+=sh__sx[(threadIdx.x+1)];
sh__sy[threadIdx.x]+=sh__sy[(threadIdx.x+1)];
}
if ((threadIdx.x==0))
{
red__sx[_bid]=sh__sx[0];
red__sy[_bid]=sh__sy[0];
}
}

int main(int argc, char *  * argv)
{
float Mops;
float t1;
float t2;
float sx;
float sy;
float tm;
float an;
float tt;
float gc;
float dum[3] =  { 1.0, 1.0, 1.0 } ;
int np;
int i;
int nit;
int k_offset;
int j;
int nthreads = 1;
int verified;
char size[(13+1)];
/* character13 */
/*

c   Because the size of the problem is too large to store in a 32-bit
c   integer for some classes, we put it into a string (for printing).
c   Have to strip off the decimal point put in there by the floating
c   point print statement (internal file)

*/
int _ret_val_0;

////////////////////////////////
// CUDA Device Initialization //
////////////////////////////////
int deviceCount;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
if (deviceCount == 0) {
	fprintf(stderr, "cutil error: no devices supporting CUDA.\n");
	exit(EXIT_FAILURE);
}
int dev = 0;
hipDeviceProp_t deviceProp;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));
	fprintf(stderr, "Using device %d: %s\n", dev, deviceProp.name);
CUDA_SAFE_CALL(hipSetDevice(dev));


float * red__sx;
float * sx__extended;
int _ti_100_0;
float * red__sy;
float * sy__extended;
float * gpu__x;
size_t pitch__x;
float * x__extended;
gpuBytes=sizeof (float);
CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__an__main)), gpuBytes));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__k_offset__main)), gpuBytes));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__l__main)), gpuBytes));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__np__main)), gpuBytes));
printf("\n\n NAS Parallel Benchmarks 2.3 OpenMP C version"" - EP Benchmark\n");
sprintf(size, "%12.0f", pow(2.0, (32+1)));
#pragma loop name main#0 
for (j=13; j>=1; j -- )
{
if ((size[j]=='.'))
{
size[j]=' ';
}
}
printf(" Number of random numbers generated: %13s\n", size);
verified=0;
/*

c   Compute the number of "batches" of random number pairs generated 
c   per processor. Adjust if the number of processors does not evenly 
c   divide the total number

*/
np=(1<<(32-1));
/*

c   Call the random number generator functions and initialize
c   the x-array to reduce the effects of paging on the timings.
c   Also, call all mathematical functions that are used. Make
c   sure these initializations cannot be eliminated as dead code.

*/
vranlc(0, ( & dum[0]), dum[1], ( & dum[2]));
dum[0]=randlc(( & dum[1]), dum[2]);
#pragma loop name main#1 
for (i=0; i<(2*(1<<1)); i ++ )
{
x[i]=( - 1.0E99);
}
Mops=log(sqrt(fabs(((1.0>1.0) ? 1.0 : 1.0))));
timer_clear(1);
timer_clear(2);
timer_clear(3);
timer_start(1);
vranlc(0, ( & t1), 1.220703125E9, x);
/*   Compute AN = A ^ (2 NK) (mod 2^46). */
t1=1.220703125E9;
#pragma loop name main#2 
for (i=1; i<=(1+1); i ++ )
{
t2=randlc(( & t1), t1);
}
an=t1;
tt=2.71828183E8;
gc=0.0;
sx=0.0;
sy=0.0;
#pragma loop name main#3 
for (i=0; i<=(10-1); i ++ )
{
q[i]=0.0;
}
/*

c   Each instance of this loop may be performed independently. We compute
c   the k offsets separately to take into account the fact that some nodes
c   have more numbers to generate than others

*/
k_offset=( - 1);
{
float t1;
float t2;
float t3;
float t4;
float x1;
float x2;
int kk;
int i;
int ik;
int l;
/* #pragma omp parallel for reduction(+:sx,sy) */
dim3 dimBlock0(gpuNumThreads, 1, 1);
gpuNumBlocks=((int)ceil((((float)np)/1024.0F)));
if ((gpuNumBlocks>MAX_GDIMENSION))
{
gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
gpuNumBlocks1=MAX_NDIMENSION;
}
else
{
gpuNumBlocks2=1;
gpuNumBlocks1=gpuNumBlocks;
}
dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
totalNumThreads=(gpuNumBlocks*gpuNumThreads);
gpuBytes=(gpuNumBlocks*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void *  * )( & red__sx)), gpuBytes));
sx__extended=((float * )malloc(gpuBytes));
gpuBytes=(gpuNumBlocks*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void *  * )( & red__sy)), gpuBytes));
sy__extended=((float * )malloc(gpuBytes));
gpuBytes=sizeof (float);
CUDA_SAFE_CALL(hipMemcpy(gpu__an__main, ( & an), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMemcpy(gpu__k_offset__main, ( & k_offset), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMemcpy(gpu__l__main, ( & l), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMemcpy(gpu__np__main, ( & np), gpuBytes, hipMemcpyHostToDevice));
CUDA_SAFE_CALL(hipMallocPitch(((void *  * )( & gpu__x)), ( & pitch__x), (totalNumThreads*sizeof (float)), (2*(1<<1))));
gpuBytes=(pitch__x*(2*(1<<1)));
x__extended=((float * )malloc(gpuBytes));
#pragma omp parallel threadprivate(x) shared(an, k_offset, l, np) private(i, ik, k, kk, t1, t2, t3, t4, x1, x2) reduction(+: sx, sy)
#pragma cuda ainfo kernelid(0) procname(main) 
#pragma cuda gpurun cudafree(an, k_offset, l, np) 
#pragma cuda gpurun nog2cmemtr(an, k_offset, np) 
main_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(red__sx, red__sy, gpu__an__main, gpu__k_offset__main, gpu__l__main, gpu__np__main, gpu__x, pitch__x);
gpuBytes=(pitch__x*(2*(1<<1)));
CUDA_SAFE_CALL(hipMemcpy2D(x__extended, (totalNumThreads*sizeof (float)), gpu__x, pitch__x, (totalNumThreads*sizeof (float)), (2*(1<<1)), hipMemcpyDeviceToHost));
for (_ti_100_0=0; _ti_100_0<(2*(1<<1)); _ti_100_0 ++ )
{
x[_ti_100_0]=( * ((float * )(((char * )x__extended)+(_ti_100_0*pitch__x))));
}
free(x__extended);
CUDA_SAFE_CALL(hipFree(gpu__x));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMemcpy(( & l), gpu__l__main, gpuBytes, hipMemcpyDeviceToHost));
gpuBytes=(gpuNumBlocks*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(sy__extended, red__sy, gpuBytes, hipMemcpyDeviceToHost));
for (_ti_100_0=0; _ti_100_0<gpuNumBlocks; _ti_100_0 ++ )
{
sy+=sy__extended[_ti_100_0];
}
free(sy__extended);
CUDA_SAFE_CALL(hipFree(red__sy));
gpuBytes=(gpuNumBlocks*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(sx__extended, red__sx, gpuBytes, hipMemcpyDeviceToHost));
for (_ti_100_0=0; _ti_100_0<gpuNumBlocks; _ti_100_0 ++ )
{
sx+=sx__extended[_ti_100_0];
}
free(sx__extended);
CUDA_SAFE_CALL(hipFree(red__sx));
}
/* end of parallel region */
#pragma loop name main#5 
for (i=0; i<=(10-1); i ++ )
{
gc=(gc+q[i]);
}
timer_stop(1);
tm=timer_read(1);
nit=0;
if ((32==24))
{
if (((fabs(((sx-( - 3247.83465203474))/sx))<=1.0E-8)&&(fabs(((sy-( - 6958.407078382297))/sy))<=1.0E-8)))
{
verified=1;
}
}
else
{
if ((32==25))
{
if (((fabs(((sx-( - 2863.319731645753))/sx))<=1.0E-8)&&(fabs(((sy-( - 6320.053679109499))/sy))<=1.0E-8)))
{
verified=1;
}
}
else
{
if ((32==28))
{
if (((fabs(((sx-( - 4295.875165629892))/sx))<=1.0E-8)&&(fabs(((sy-( - 15807.32573678431))/sy))<=1.0E-8)))
{
verified=1;
}
}
else
{
if ((32==30))
{
if (((fabs(((sx-40338.15542441498)/sx))<=1.0E-8)&&(fabs(((sy-( - 26606.69192809235))/sy))<=1.0E-8)))
{
verified=1;
}
}
else
{
if ((32==32))
{
if (((fabs(((sx-47643.67927995374)/sx))<=1.0E-8)&&(fabs(((sy-( - 80840.72988043731))/sy))<=1.0E-8)))
{
verified=1;
}
}
}
}
}
}
Mops=((pow(2.0, (32+1))/tm)/1000000.0);
printf("EP Benchmark Results: \n""CPU Time = %10.4f\n""N = 2^%5d\n""No. Gaussian Pairs = %15.0f\n""Sums = %25.15e %25.15e\n""Counts:\n", tm, 32, gc, sx, sy);
#pragma loop name main#6 
for (i=0; i<=(10-1); i ++ )
{
printf("%3d %15.0f\n", i, q[i]);
}
c_print_results("EP", 'C', (32+1), 0, 0, nit, nthreads, tm, Mops, "Random numbers generated", verified, "2.3", "23 Aug 2012", "gcc", "gcc", "-lm", "-I../common", "-O3 ", "(none)", "randdp");
if ((0==1))
{
printf("Total time:     %f", timer_read(1));
printf("Gaussian pairs: %f", timer_read(2));
printf("Random numbers: %f", timer_read(3));
}
printf("/***********************/ \n/* Input Configuration */ \n/***********************/ \n");
printf("====> GPU Block Size: 1024 \n");
printf("/**********************/ \n/* Used Optimizations */ \n/**********************/ \n");
printf("====> MatrixTranspose Opt is used.\n");
printf("====> ParallelLoopSwap Opt is used.\n");
printf("====> LoopCollapse Opt is used.\n");
printf("====> Unrolling-on-reduction Opt is used.\n");
printf("====> Allocate GPU variables as global ones.\n");
printf("====> CPU-GPU Mem Transfer Opt Level: 4\n");
printf("====> Cuda Malloc Opt Level: 1\n");
printf("====> Assume that all loops have non-zero iterations.\n");
printf("====> Cache shared array elements onto GPU registers.\n");
printf("====> local array reduction variable configuration = 1\n");
CUDA_SAFE_CALL(hipFree(gpu__an__main));
CUDA_SAFE_CALL(hipFree(gpu__k_offset__main));
CUDA_SAFE_CALL(hipFree(gpu__l__main));
CUDA_SAFE_CALL(hipFree(gpu__np__main));
fflush(stdout);
fflush(stderr);
return _ret_val_0;
}

