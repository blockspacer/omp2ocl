#include "hip/hip_runtime.h"
/*
   --------------------------------------------------------------------

   NAS Parallel Benchmarks 2.3 OpenMP C versions - FT

   This benchmark is an OpenMP C version of the NPB FT code.

   The OpenMP C versions are developed by RWCP and derived from the serial
   Fortran versions in "NPB 2.3-serial" developed by NAS.

   Permission to use, copy, distribute and modify this software for any
   purpose with or without fee is hereby granted.
   This software is provided "as is" without express or implied warranty.

   Send comments on the OpenMP C versions to pdp-openmp@rwcp.or.jp

   Information on OpenMP activities at RWCP is available at:

http:pdplab.trc.rwcp.or.jppdperf/Omni/

Information on NAS Parallel Benchmarks 2.3 is available at:

http:www.nas.nasa.gov/NAS/NPB/

--------------------------------------------------------------------
 */
/*
   --------------------------------------------------------------------

Authors: D. Bailey
W. Saphir

OpenMP C version: S. Satoh

--------------------------------------------------------------------
 */
#include "npb-C.h"
/* global variables */
#include "global.h"

#ifndef __O2G_HEADER__ 

#define __O2G_HEADER__ 

/******************************************/
/* Added codes for OpenMP2GPU translation */
/******************************************/
#include <cutil.h>
#include <math.h>
#define MAX(a,b) (((a) > (b)) ? (a) : (b))


static int gpuNumThreads = BLOCK_SIZE;
static int gpuNumBlocks;
static int gpuNumBlocks1;
static int gpuNumBlocks2;
static int totalNumThreads;
unsigned int gpuGmemSize = 0;
unsigned int gpuSmemSize = 0;
static unsigned int gpuBytes = 0;

#endif 
/* End of __O2G_HEADER__ */



int * gpu__fftblock;
int * gpu__fftblockpad;
double * gpu__u_imag;
double * gpu__u_real;
double * gpu__u1_imag__main;
double * gpu__u1_real__main;
double * gpu__u0_imag__main;
double * gpu__u0_real__main;
double * gpu__u2_imag__main;
double * gpu__u2_real__main;
int * gpu__xend;
int * gpu__xstart;
int * gpu__yend;
int * gpu__ystart;
int * gpu__zend;
int * gpu__zstart;
int * gpu__dims;
size_t pitch__dims;
int * gpu__indexmap__main;
double * gpu__ex;
static double tmp__compute_initial_conditions[(((512*2)*512)+1)];
static double yy0_real[512][18];
static double yy0_imag[512][18];
static double yy1_real[512][18];
static double yy1_imag[512][18];
#pragma omp threadprivate(yy0_real)
#pragma omp threadprivate(yy0_imag)
#pragma omp threadprivate(yy1_real)
#pragma omp threadprivate(yy1_imag)
/* function declarations */
static void evolve_cloned0(double u0_real[256][256][512], double u0_imag[256][256][512], double u1_real[256][256][512], double u1_imag[256][256][512], int t, int indexmap[256][256][512], int d[3]);
static void compute_initial_conditions(double u0_real[256][256][512], double u0_imag[256][256][512], int d[3]);
static void ipow46(double a, int exponent, double * result);
static void setup(void );
static void compute_indexmap(int indexmap[256][256][512], int d[3]);
static void compute_indexmap_clnd1(int indexmap[256][256][512], int d[3]);
static void print_timers(void );
static void fft(int dir, double x1_real[256][256][512], double x1_imag[256][256][512], double x2_real[256][256][512], double x2_imag[256][256][512]);
static void fft_clnd2_cloned0(int dir, double x1_real[256][256][512], double x1_imag[256][256][512], double x2_real[256][256][512], double x2_imag[256][256][512]);
static void fft_clnd1(int dir, double x1_real[256][256][512], double x1_imag[256][256][512], double x2_real[256][256][512], double x2_imag[256][256][512]);
static void cffts1(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts1_clnd5(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts1_clnd4(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts1_clnd3_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts1_clnd2_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts1_clnd1(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts2(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts2_clnd5(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts2_clnd4(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts2_clnd3_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts2_clnd2_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts2_clnd1(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts3(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts3_clnd5(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts3_clnd4(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts3_clnd3_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts3_clnd2_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void cffts3_clnd1(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18]);
static void fft_init(int n);
__device__ static void dev_cfftz(int is, int m, int n, double x_real[][512][18], double x_imag[][512][18], double y_real[][512][18], double y_imag[][512][18], int * fftblock, int * fftblockpad, double u_imag[512], double u_real[512], int _gtid);
__device__ static void dev_fftz2(int is, int l, int m, int n, int ny, int ny1, double u_real[512], double u_imag[512], double x_real[][512][18], double x_imag[][512][18], double y_real[][512][18], double y_imag[][512][18], int _gtid);
static int ilog2(int n);
static void checksum(int i, double u1_real[256][256][512], double u1_imag[256][256][512], int d[3]);
static void verify(int d1, int d2, int d3, int nt, int * verified, char * cclass);
/*  */
/*          E  L  A  P  S  E  D  _  T  I  M  E */
/*  */
double elapsed_time(void )
{
	double t;
	wtime(( & t));
	return t;
}

double start[64];
double elapsed[64];
/*  */
/*             T  I  M  E  R  _  C  L  E  A  R */
/*  */
void timer_clear(int n)
{
	elapsed[n]=0.0;
	return ;
}

/*  */
/*             T  I  M  E  R  _  S  T  A  R  T */
/*  */
void timer_start(int n)
{
	start[n]=elapsed_time();
	return ;
}

/*  */
/*             T  I  M  E  R  _  S  T  O  P */
/*  */
void timer_stop(int n)
{
	double t;
	double now;
	now=elapsed_time();
	t=(now-start[n]);
	elapsed[n]+=t;
	return ;
}

/*  */
/*             T  I  M  E  R  _  R  E  A  D */
/*  */
double timer_read(int n)
{
	double _ret_val_0;
	_ret_val_0=elapsed[n];
	return _ret_val_0;
}

static void c_print_results(char * name, char cccccclass, int n1, int n2, int n3, int niter, int nthreads, double t, double mops, char * optype, int passed_verification, char * npbversion, char * compiletime, char * cc, char * clink, char * c_lib, char * c_inc, char * cflags, char * clinkflags, char * rand)
{
	printf("\n\n %s Benchmark Completed\n", name);
	printf(" Class           =                        %c\n", cccccclass);
	/* as in IS */
	if (((n2==0)&&(n3==0)))
	{
		printf(" Size            =             %12d\n", n1);
	}
	else
	{
		printf(" Size            =              %3dx%3dx%3d\n", n1, n2, n3);
	}
	printf(" Iterations      =             %12d\n", niter);
	printf(" Threads         =             %12d\n", nthreads);
	printf(" Time in seconds =             %12.2f\n", t);
	printf(" Mop/s total     =             %12.2f\n", mops);
	printf(" Operation type  = %24s\n", optype);
	if (passed_verification)
	{
		printf(" Verification    =               SUCCESSFUL\n");
	}
	else
	{
		printf(" Verification    =             UNSUCCESSFUL\n");
	}
	printf(" Version         =             %12s\n", npbversion);
	printf(" Compile date    =             %12s\n", compiletime);
	printf("\n Compile options:\n");
	printf("    CC           = %s\n", cc);
	printf("    CLINK        = %s\n", clink);
	printf("    C_LIB        = %s\n", c_lib);
	printf("    C_INC        = %s\n", c_inc);
	printf("    CFLAGS       = %s\n", cflags);
	printf("    CLINKFLAGS   = %s\n", clinkflags);
	printf("    RAND         = %s\n", rand);
	/*
	   printf( "\n\n" );
	   printf( " Please send the results of this run to:\n\n" );
	   printf( " NPB Development Team\n" );
	   printf( " Internet: npb@nas.nasa.gov\n \n" );
	   printf( " If email is not available, send this to:\n\n" );
	   printf( " MS T27A-1\n" );
	   printf( " NASA Ames Research Center\n" );
	   printf( " Moffett Field, CA  94035-1000\n\n" );
	   printf( " Fax: 415-604-3957\n\n" );
	 */
	return ;
}

/*
   c---------------------------------------------------------------------
   c---------------------------------------------------------------------
 */
double randlc(double * x, double a)
{
	/*
	   c---------------------------------------------------------------------
	   c---------------------------------------------------------------------
	 */
	/*
	   c---------------------------------------------------------------------
	   c
	   c   This routine returns a uniform pseudorandom double precision number in the
	   c   range (0, 1) by using the linear congruential generator
	   c
	   c   x_{k+1} = a x_k  (mod 2^46)
	   c
	   c   where 0 < x_k < 2^46 and 0 < a < 2^46.  This scheme generates 2^44 numbers
	   c   before repeating.  The argument A is the same as 'a' in the above formula,
	   c   and X is the same as x_0.  A and X must be odd double precision integers
	   c   in the range (1, 2^46).  The returned value RANDLC is normalized to be
	   c   between 0 and 1, i.e. RANDLC = 2^(-46) x_1.  X is updated to contain
	   c   the new seed x_1, so that subsequent calls to RANDLC using the same
	   c   arguments will generate a continuous sequence.
	   c
	   c   This routine should produce the same results on any computer with at least
	   c   48 mantissa bits in double precision floating point data.  On 64 bit
	   c   systems, double precision should be disabled.
	   c
	   c   David H. Bailey     October 26, 1990
	   c
	   c---------------------------------------------------------------------
	 */
	double t1;
	double t2;
	double t3;
	double t4;
	double a1;
	double a2;
	double x1;
	double x2;
	double z;
	/*
	   c---------------------------------------------------------------------
	   c   Break A into two parts such that A = 2^23 A1 + A2.
	   c---------------------------------------------------------------------
	 */
	double _ret_val_0;
	t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*a);
	a1=((int)t1);
	a2=(a-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*a1));
	/*
	   c---------------------------------------------------------------------
	   c   Break X into two parts such that X = 2^23 X1 + X2, compute
	   c   Z = A1 * X2 + A2 * X1  (mod 2^23), and then
	   c   X = 2^23 * Z + A2 * X2  (mod 2^46).
	   c---------------------------------------------------------------------
	 */
	t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*( * x));
	x1=((int)t1);
	x2=(( * x)-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*x1));
	t1=((a1*x2)+(a2*x1));
	t2=((int)(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*t1));
	z=(t1-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*t2));
	t3=((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*z)+(a2*x2));
	t4=((int)((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*t3));
	( * x)=(t3-((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0))*t4));
	_ret_val_0=((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*( * x));
	return _ret_val_0;
}

/*
   c---------------------------------------------------------------------
   c---------------------------------------------------------------------
 */
void vranlc(int n, double * x_seed, double a, double y[])
{
	/*
	   c---------------------------------------------------------------------
	   c---------------------------------------------------------------------
	 */
	/*
	   c---------------------------------------------------------------------
	   c
	   c   This routine generates N uniform pseudorandom double precision numbers in
	   c   the range (0, 1) by using the linear congruential generator
	   c
	   c   x_{k+1} = a x_k  (mod 2^46)
	   c
	   c   where 0 < x_k < 2^46 and 0 < a < 2^46.  This scheme generates 2^44 numbers
	   c   before repeating.  The argument A is the same as 'a' in the above formula,
	   c   and X is the same as x_0.  A and X must be odd double precision integers
	   c   in the range (1, 2^46).  The N results are placed in Y and are normalized
	   c   to be between 0 and 1.  X is updated to contain the new seed, so that
	   c   subsequent calls to VRANLC using the same arguments will generate a
	   c   continuous sequence.  If N is zero, only initialization is performed, and
	   c   the variables X, A and Y are ignored.
	   c
	   c   This routine is the standard version designed for scalar or RISC systems.
	   c   However, it should produce the same results on any single processor
	   c   computer with at least 48 mantissa bits in double precision floating point
	   c   data.  On 64 bit systems, double precision should be disabled.
	   c
	   c---------------------------------------------------------------------
	 */
	int i;
	double x;
	double t1;
	double t2;
	double t3;
	double t4;
	double a1;
	double a2;
	double x1;
	double x2;
	double z;
	/*
	   c---------------------------------------------------------------------
	   c   Break A into two parts such that A = 2^23 A1 + A2.
	   c---------------------------------------------------------------------
	 */
	t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*a);
	a1=((int)t1);
	a2=(a-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*a1));
	x=( * x_seed);
	/*
	   c---------------------------------------------------------------------
	   c   Generate N results.   This loop is not vectorizable.
	   c---------------------------------------------------------------------
	 */
#pragma loop name vranlc#0 
	for (i=1; i<=n; i ++ )
	{
		/*
		   c---------------------------------------------------------------------
		   c   Break X into two parts such that X = 2^23 X1 + X2, compute
		   c   Z = A1 * X2 + A2 * X1  (mod 2^23), and then
		   c   X = 2^23 * Z + A2 * X2  (mod 2^46).
		   c---------------------------------------------------------------------
		 */
		t1=(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*x);
		x1=((int)t1);
		x2=(x-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*x1));
		t1=((a1*x2)+(a2*x1));
		t2=((int)(((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*t1));
		z=(t1-(((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*t2));
		t3=((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*z)+(a2*x2));
		t4=((int)((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*t3));
		x=(t3-((((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*((((((((((((((((((((((2.0*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0)*2.0))*t4));
		y[i]=((((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*((((((((((((((((((((((0.5*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5)*0.5))*x);
	}
	( * x_seed)=x;
	return ;
}

int main(int argc, char *  * argv)
{
	/*
	   c-------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int i;
	/*
	   ------------------------------------------------------------------
	   c u0, u1, u2 are the main arrays in the problem. 
	   c Depending on the decomposition, these arrays will have different 
	   c dimensions. To accomodate all possibilities, we allocate them as 
	   c one-dimensional arrays and pass them to subroutines for different 
	   c views
	   c  - u0 contains the initial (transformed) initial condition
	   c  - u1 and u2 are working arrays
	   c  - indexmap maps i,j,k of u0 to the correct i^2+j^2+k^2 for the
	   c    time evolution operator. 
	   c-----------------------------------------------------------------
	 */
	/*
	   --------------------------------------------------------------------
	   c Large arrays are in common so that they are allocated on the
	   c heap rather than the stack. This common block is not
	   c referenced directly anywhere else. Padding is to avoid accidental 
	   c cache problems, since all array sizes are powers of two.
	   c-------------------------------------------------------------------
	 */
	static double u0_real[256][256][512];
	static double u0_imag[256][256][512];
	static double u1_real[256][256][512];
	static double u1_imag[256][256][512];
	static double u2_real[256][256][512];
	static double u2_imag[256][256][512];
	static int indexmap[256][256][512];
	int iter;
	int nthreads = 1;
	double total_time;
	double mflops;
	int verified;
	char cclass;
	/*
	   --------------------------------------------------------------------
	   c Run the entire problem once to make sure all data is touched. 
	   c This reduces variable startup costs, which is important for such a 
	   c short benchmark. The other NPB 2 implementations are similar. 
	   c-------------------------------------------------------------------
	 */
	int _ret_val_0;

	////////////////////////////////
	// CUDA Device Initialization //
	////////////////////////////////
	int deviceCount;
	CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0) {
		fprintf(stderr, "cutil error: no devices supporting CUDA.\n");
		exit(EXIT_FAILURE);
	}
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));
	fprintf(stderr, "Using device %d: %s\n", dev, deviceProp.name);
	CUDA_SAFE_CALL(hipSetDevice(dev));


	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__fftblock)), gpuBytes));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__fftblockpad)), gpuBytes));
	gpuBytes=(512*sizeof (double));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__u_imag)), gpuBytes));
	gpuBytes=(512*sizeof (double));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__u_real)), gpuBytes));
	gpuBytes=(((256*256)*512)*sizeof (double));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__u1_imag__main)), gpuBytes));
	gpuBytes=(((256*256)*512)*sizeof (double));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__u1_real__main)), gpuBytes));
	gpuBytes=(((256*256)*512)*sizeof (double));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__u0_imag__main)), gpuBytes));
	gpuBytes=(((256*256)*512)*sizeof (double));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__u0_real__main)), gpuBytes));
	gpuBytes=(((256*256)*512)*sizeof (double));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__u2_imag__main)), gpuBytes));
	gpuBytes=(((256*256)*512)*sizeof (double));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__u2_real__main)), gpuBytes));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__xend)), gpuBytes));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__xstart)), gpuBytes));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yend)), gpuBytes));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__ystart)), gpuBytes));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__zend)), gpuBytes));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__zstart)), gpuBytes));
	CUDA_SAFE_CALL(hipMallocPitch(((void *  * )( & gpu__dims)), ( & pitch__dims), (3*sizeof (int)), 3));
	gpuBytes=(((256*256)*512)*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__indexmap__main)), gpuBytes));
	gpuBytes=(((20*((((512*512)/4)+((256*256)/4))+((256*256)/4)))+1)*sizeof (double));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__ex)), gpuBytes));
#pragma loop name main#0 
	for (i=0; i<7; i ++ )
	{
		timer_clear(i);
	}
	setup();
	/* #pragma omp parallel */
	{
		compute_indexmap(indexmap, dims[2]);
		/* #pragma omp single */
		{
			compute_initial_conditions(u1_real, u1_imag, dims[0]);
			fft_init(dims[0][0]);
		}
		fft(1, u1_real, u1_imag, u0_real, u0_imag);
	}
	/* end parallel */
	/*
	   --------------------------------------------------------------------
	   c Start over from the beginning. Note that all operations must
	   c be timed, in contrast to other benchmarks. 
	   c-------------------------------------------------------------------
	 */
#pragma loop name main#1 
	for (i=0; i<7; i ++ )
	{
		timer_clear(i);
	}
	timer_start(0);
	if ((0==1))
	{
		timer_start(1);
	}
	/* #pragma omp parallel private(iter) firstprivate(niter) */
	{
		compute_indexmap_clnd1(indexmap, dims[2]);
		/* #pragma omp single */
		{
			compute_initial_conditions(u1_real, u1_imag, dims[0]);
			fft_init(dims[0][0]);
		}
		if ((0==1))
		{
			/* #pragma omp master */
			timer_stop(1);
		}
		if ((0==1))
		{
			/* #pragma omp master    */
			timer_start(2);
		}
		fft_clnd1(1, u1_real, u1_imag, u0_real, u0_imag);
		if ((0==1))
		{
			/* #pragma omp master       */
			timer_stop(2);
		}
#pragma loop name main#2 
		for (iter=1; iter<=niter; iter ++ )
		{
			if ((0==1))
			{
				/* #pragma omp master       */
				timer_start(3);
			}
			evolve_cloned0(u0_real, u0_imag, u1_real, u1_imag, iter, indexmap, dims[0]);
			if ((0==1))
			{
				/* #pragma omp master       */
				timer_stop(3);
			}
			if ((0==1))
			{
				/* #pragma omp master       */
				timer_start(2);
			}
			fft_clnd2_cloned0(( - 1), u1_real, u1_imag, u2_real, u2_imag);
			if ((0==1))
			{
				/* #pragma omp master       */
				timer_stop(2);
			}
			if ((0==1))
			{
				/* #pragma omp master       */
				timer_start(4);
			}
			checksum(iter, u2_real, u2_imag, dims[0]);
			if ((0==1))
			{
				/* #pragma omp master       */
				timer_stop(4);
			}
		}
		/* #pragma omp single */
		verify(512, 256, 256, niter, ( & verified), ( & cclass));
	}
	/* end parallel */
	timer_stop(0);
	total_time=timer_read(0);
	if ((total_time!=0.0))
	{
		mflops=(((1.0E-6*((double)33554432))*((14.8157+(7.19641*log(((double)33554432))))+((5.23518+(7.21113*log(((double)33554432))))*niter)))/total_time);
	}
	else
	{
		mflops=0.0;
	}
	c_print_results("FT", cclass, 512, 256, 256, niter, nthreads, total_time, mflops, "          floating point", verified, "2.3", "20 Feb 2012", "gcc", "gcc", "-lm", "-I../common", "-O3 ", "(none)", "randdp");
	if ((0==1))
	{
		print_timers();
	}
	printf("/***********************/ \n/* Input Configuration */ \n/***********************/ \n");
	printf("====> GPU Block Size: 1024 \n");
	printf("/**********************/ \n/* Used Optimizations */ \n/**********************/ \n");
	printf("====> MallocPitch Opt is used.\n");
	printf("====> MatrixTranspose Opt is used.\n");
	printf("====> ParallelLoopSwap Opt is used.\n");
	printf("====> LoopCollapse Opt is used.\n");
	printf("====> Unrolling-on-reduction Opt is used.\n");
	printf("====> Allocate GPU variables as global ones.\n");
	printf("====> Optimize globally allocated GPU variables .\n");
	printf("====> CPU-GPU Mem Transfer Opt Level: 4\n");
	printf("====> Cuda Malloc Opt Level: 1\n");
	printf("====> Assume that all loops have non-zero iterations.\n");
	printf("====> Cache shared scalar variables onto GPU registers.\n");
	printf("====> Cache shared array elements onto GPU registers.\n");
	printf("====> Cache private array variables onto GPU shared memory.\n");
	printf("====> local array reduction variable configuration = 1\n");
	CUDA_SAFE_CALL(hipFree(gpu__fftblock));
	CUDA_SAFE_CALL(hipFree(gpu__fftblockpad));
	CUDA_SAFE_CALL(hipFree(gpu__u_imag));
	CUDA_SAFE_CALL(hipFree(gpu__u_real));
	CUDA_SAFE_CALL(hipFree(gpu__u1_imag__main));
	CUDA_SAFE_CALL(hipFree(gpu__u1_real__main));
	CUDA_SAFE_CALL(hipFree(gpu__u0_imag__main));
	CUDA_SAFE_CALL(hipFree(gpu__u0_real__main));
	CUDA_SAFE_CALL(hipFree(gpu__u2_imag__main));
	CUDA_SAFE_CALL(hipFree(gpu__u2_real__main));
	CUDA_SAFE_CALL(hipFree(gpu__xend));
	CUDA_SAFE_CALL(hipFree(gpu__xstart));
	CUDA_SAFE_CALL(hipFree(gpu__yend));
	CUDA_SAFE_CALL(hipFree(gpu__ystart));
	CUDA_SAFE_CALL(hipFree(gpu__zend));
	CUDA_SAFE_CALL(hipFree(gpu__zstart));
	CUDA_SAFE_CALL(hipFree(gpu__dims));
	CUDA_SAFE_CALL(hipFree(gpu__indexmap__main));
	CUDA_SAFE_CALL(hipFree(gpu__ex));
	fflush(stdout);
	fflush(stderr);
	return _ret_val_0;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
__global__ void evolve_cloned0_kernel0(int * d, double * ex, int indexmap[256][256][512], int * t, double u0_imag[256][256][512], double u0_real[256][256][512], double u1_imag[256][256][512], double u1_real[256][256][512])
{
	double ex_0;
	int t_0;
	int i;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	i=_gtid;
	t_0=( * t);
	if (i<d[0])
	{
#pragma loop name evolve#0#0 
		for (j=0; j<d[1]; j ++ )
		{
#pragma loop name evolve#0#0#0 
			for (k=0; k<d[2]; k ++ )
			{
				ex_0=ex[(t_0*indexmap[k][j][i])];
				u1_real[k][j][i]=(u0_real[k][j][i]*ex_0);
				u1_imag[k][j][i]=(u0_imag[k][j][i]*ex_0);
			}
		}
	}
}

static void evolve_cloned0(double u0_real[256][256][512], double u0_imag[256][256][512], double u1_real[256][256][512], double u1_imag[256][256][512], int t, int indexmap[256][256][512], int d[3])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/*
	   --------------------------------------------------------------------
	   c evolve u0 -> u1 (t time steps) in fourier space
	   c-------------------------------------------------------------------
	 */
	int * gpu__d;
	int * gpu__t;
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[0])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=(((20*((((512*512)/4)+((256*256)/4))+((256*256)/4)))+1)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(gpu__ex, ex, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__t)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__t, ( & t), gpuBytes, hipMemcpyHostToDevice));
#pragma omp parallel for shared(d, ex, indexmap, t, u0_imag, u0_real, u1_imag, u1_real) private(i, j, k)
#pragma cuda gpurun noc2gmemtr(indexmap, u0_imag, u0_real, u1_imag, u1_real) 
#pragma cuda gpurun nocudamalloc(indexmap, u0_imag, u0_real, u1_imag, u1_real) 
#pragma cuda gpurun nocudafree(ex, indexmap, u0_imag, u0_real, u1_imag, u1_real) 
#pragma cuda gpurun multisrccg(ex) 
#pragma cuda gpurun nog2cmemtr(d, ex, indexmap, t, u0_imag, u0_real, u1_imag, u1_real) 
#pragma cuda ainfo kernelid(0) procname(evolve_cloned0) 
#pragma cuda gpurun registerRO(ex[(t*indexmap[k][j][i])], t) 
#pragma cuda gpurun cudafree(d, t) 
	evolve_cloned0_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__ex, ((int (*)[256][512])gpu__indexmap__main), gpu__t, ((double (*)[256][512])gpu__u0_imag__main), ((double (*)[256][512])gpu__u0_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__t));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
static void compute_initial_conditions(double u0_real[256][256][512], double u0_imag[256][256][512], int d[3])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/*
	   --------------------------------------------------------------------
	   c Fill in array u0 with initial conditions from 
	   c random number generator 
	   c-------------------------------------------------------------------
	 */
	int k;
	double x0;
	double start;
	double an;
	double dummy;
	int i;
	int j;
	int t;
	start=3.14159265E8;
	/*
	   --------------------------------------------------------------------
	   c Jump to the starting element for our first plane.
	   c-------------------------------------------------------------------
	 */
	ipow46(1.220703125E9, (((((zstart[0]-1)*2)*512)*256)+(((ystart[0]-1)*2)*512)), ( & an));
	dummy=randlc(( & start), an);
	ipow46(1.220703125E9, ((2*512)*256), ( & an));
	/*
	   --------------------------------------------------------------------
	   c Go through by z planes filling in one square at a time.
	   c-------------------------------------------------------------------
	 */
#pragma loop name compute_initial_conditions#0 
	for (k=0; k<dims[0][2]; k ++ )
	{
		x0=start;
		vranlc(((2*512)*dims[0][1]), ( & x0), 1.220703125E9, tmp__compute_initial_conditions);
		t=1;
#pragma loop name compute_initial_conditions#0#0 
		for (j=0; j<dims[0][1]; j ++ )
		{
#pragma loop name compute_initial_conditions#0#0#0 
			for (i=0; i<512; i ++ )
			{
				u0_real[k][j][i]=tmp__compute_initial_conditions[(t ++ )];
				u0_imag[k][j][i]=tmp__compute_initial_conditions[(t ++ )];
			}
		}
		if ((k!=dims[0][2]))
		{
			dummy=randlc(( & start), an);
		}
	}
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
static void ipow46(double a, int exponent, double * result)
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/*
	   --------------------------------------------------------------------
	   c compute a^exponent mod 2^46
	   c-------------------------------------------------------------------
	 */
	double dummy;
	double q;
	double r;
	int n;
	int n2;
	/*
	   --------------------------------------------------------------------
	   c Use
	   c   a^n = a^(n2)*a^(n/2) if n even else
	   c   a^n = a*a^(n-1)       if n odd
	   c-------------------------------------------------------------------
	 */
	( * result)=1;
	if ((exponent==0))
	{
		return ;
	}
	q=a;
	r=1;
	n=exponent;
	while (n>1)
	{
		n2=(n/2);
		if (((n2*2)==n))
		{
			dummy=randlc(( & q), q);
			n=n2;
		}
		else
		{
			dummy=randlc(( & r), q);
			n=(n-1);
		}
	}
	dummy=randlc(( & r), q);
	( * result)=r;
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
static void setup(void )
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int i;
	printf("\n\n NAS Parallel Benchmarks 2.3 OpenMP C version"" - FT Benchmark\n\n");
	niter=20;
	printf(" Size                : %3dx%3dx%3d\n", 512, 256, 256);
	printf(" Iterations          :     %7d\n", niter);
	/*
	   1004 format(' Number of processes :     ', i7)
	   1005 format(' Processor array     :     ', i3, 'x', i3)
	   1006 format(' WARNING: compiled for ', i5, ' processes. ',
	   >       ' Will not verify. ')
	 */
#pragma loop name setup#0 
	for (i=0; i<3; i ++ )
	{
		dims[i][0]=512;
		dims[i][1]=256;
		dims[i][2]=256;
	}
#pragma loop name setup#1 
	for (i=0; i<3; i ++ )
	{
		xstart[i]=1;
		xend[i]=512;
		ystart[i]=1;
		yend[i]=256;
		zstart[i]=1;
		zend[i]=256;
	}
	/*
	   --------------------------------------------------------------------
	   c Set up info for blocking of ffts and transposes.  This improves
	   c performance on cache-based systems. Blocking involves
	   c working on a chunk of the problem at a time, taking chunks
	   c along the first, second, or third dimension. 
	   c
	   c - In cffts1 blocking is on 2nd dimension (with fft on 1st dim)
	   c - In cffts23 blocking is on 1st dimension (with fft on 2nd and 3rd dims)

	   c Since 1st dim is always in processor, we'll assume it's long enough 
	   c (default blocking factor is 16 so min size for 1st dim is 16)
	   c The only case we have to worry about is cffts1 in a 2d decomposition. 
	   c so the blocking factor should not be larger than the 2nd dimension. 
	   c-------------------------------------------------------------------
	 */
	fftblock=16;
	fftblockpad=18;
	if ((fftblock!=16))
	{
		fftblockpad=(fftblock+3);
	}
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
__global__ void compute_indexmap_kernel0(int * dims, size_t pitch__dims, int indexmap[256][256][512], int * xstart_i, int * ystart_i, int * zstart_i)
{
	int i;
	int ii;
	int ii2;
	int ij2;
	int j;
	int jj;
	int k;
	int kk;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	i=_gtid;
	if (i<( * (((int * )(((char * )dims)+(2*pitch__dims)))+0)))
	{
#pragma loop name compute_indexmap#0#0 
		for (j=0; j<( * (((int * )(((char * )dims)+(2*pitch__dims)))+1)); j ++ )
		{
#pragma loop name compute_indexmap#0#0#0 
			for (k=0; k<( * (((int * )(((char * )dims)+(2*pitch__dims)))+2)); k ++ )
			{
				ii=((((((i+1)+( * xstart_i))-2)+(512/2))%512)-(512/2));
				ii2=(ii*ii);
				jj=((((((j+1)+( * ystart_i))-2)+(256/2))%256)-(256/2));
				ij2=((jj*jj)+ii2);
				kk=((((((k+1)+( * zstart_i))-2)+(256/2))%256)-(256/2));
				indexmap[k][j][i]=((kk*kk)+ij2);
			}
		}
	}
}

static void compute_indexmap(int indexmap[256][256][512], int d[3])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/*
	   --------------------------------------------------------------------
	   c compute function from local (i,j,k) to ibar^2+jbar^2+kbar^2 
	   c for time evolution exponent. 
	   c-------------------------------------------------------------------
	 */
	int i;
	double ap;
	int xstart_i;
	int ystart_i;
	int zstart_i;
	/*
	   --------------------------------------------------------------------
	   c basically we want to convert the fortran indices 
	   c   1 2 3 4 5 6 7 8 
	   c to 
	   c   0 1 2 3 -4 -3 -2 -1
	   c The following magic formula does the trick:
	   c mod(i-1+n2, n) - n/2
	   c-------------------------------------------------------------------
	 */
	int * gpu__xstart_i;
	int * gpu__ystart_i;
	int * gpu__zstart_i;
	xstart_i=xstart[2];
	ystart_i=ystart[2];
	zstart_i=zstart[2];
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)dims[2][0])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	CUDA_SAFE_CALL(hipMemcpy2D(gpu__dims, pitch__dims, dims, (3*sizeof (int)), (3*sizeof (int)), 3, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__xstart_i)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__xstart_i, ( & xstart_i), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__ystart_i)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__ystart_i, ( & ystart_i), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__zstart_i)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__zstart_i, ( & zstart_i), gpuBytes, hipMemcpyHostToDevice));
#pragma omp parallel for shared(dims, indexmap, xstart_i, ystart_i, zstart_i) private(i, ii, ii2, ij2, j, jj, k, kk) schedule(static)
#pragma cuda gpurun nocudafree(dims, indexmap) 
#pragma cuda gpurun nog2cmemtr(dims, indexmap, xstart_i, ystart_i, zstart_i) 
#pragma cuda ainfo kernelid(0) procname(compute_indexmap) 
#pragma cuda gpurun cudafree(xstart_i, ystart_i, zstart_i) 
#pragma cuda gpurun noc2gmemtr(indexmap) 
	compute_indexmap_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__dims, pitch__dims, ((int (*)[256][512])gpu__indexmap__main), gpu__xstart_i, gpu__ystart_i, gpu__zstart_i);
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__zstart_i));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__ystart_i));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__xstart_i));
	/*
	   --------------------------------------------------------------------
	   c compute array of exponentials for time evolution. 
	   c-------------------------------------------------------------------
	 */
	/* #pragma omp single */
	{
		ap=(((( - 4.0)*1.0E-6)*3.141592653589793)*3.141592653589793);
		ex[0]=1.0;
		ex[1]=exp(ap);
#pragma loop name compute_indexmap#1 
		for (i=2; i<=(20*((((512*512)/4)+((256*256)/4))+((256*256)/4))); i ++ )
		{
			ex[i]=(ex[(i-1)]*ex[1]);
		}
	}
	/* end single */
	return ;
}

__global__ void compute_indexmap_clnd1_kernel0(int * dims, size_t pitch__dims, int indexmap[256][256][512], int * xstart_i, int * ystart_i, int * zstart_i)
{
	int i;
	int ii;
	int ii2;
	int ij2;
	int j;
	int jj;
	int k;
	int kk;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	i=_gtid;
	if (i<( * (((int * )(((char * )dims)+(2*pitch__dims)))+0)))
	{
#pragma loop name compute_indexmap#0#0 
		for (j=0; j<( * (((int * )(((char * )dims)+(2*pitch__dims)))+1)); j ++ )
		{
#pragma loop name compute_indexmap#0#0#0 
			for (k=0; k<( * (((int * )(((char * )dims)+(2*pitch__dims)))+2)); k ++ )
			{
				ii=((((((i+1)+( * xstart_i))-2)+(512/2))%512)-(512/2));
				ii2=(ii*ii);
				jj=((((((j+1)+( * ystart_i))-2)+(256/2))%256)-(256/2));
				ij2=((jj*jj)+ii2);
				kk=((((((k+1)+( * zstart_i))-2)+(256/2))%256)-(256/2));
				indexmap[k][j][i]=((kk*kk)+ij2);
			}
		}
	}
}

static void compute_indexmap_clnd1(int indexmap[256][256][512], int d[3])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/*
	   --------------------------------------------------------------------
	   c compute function from local (i,j,k) to ibar^2+jbar^2+kbar^2 
	   c for time evolution exponent. 
	   c-------------------------------------------------------------------
	 */
	int i;
	double ap;
	int xstart_i;
	int ystart_i;
	int zstart_i;
	/*
	   --------------------------------------------------------------------
	   c basically we want to convert the fortran indices 
	   c   1 2 3 4 5 6 7 8 
	   c to 
	   c   0 1 2 3 -4 -3 -2 -1
	   c The following magic formula does the trick:
	   c mod(i-1+n2, n) - n/2
	   c-------------------------------------------------------------------
	 */
	int * gpu__xstart_i;
	int * gpu__ystart_i;
	int * gpu__zstart_i;
	xstart_i=xstart[2];
	ystart_i=ystart[2];
	zstart_i=zstart[2];
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__xstart_i)), gpuBytes));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)dims[2][0])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	CUDA_SAFE_CALL(hipMemcpy(gpu__xstart_i, ( & xstart_i), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__ystart_i)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__ystart_i, ( & ystart_i), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__zstart_i)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__zstart_i, ( & zstart_i), gpuBytes, hipMemcpyHostToDevice));
#pragma omp parallel for shared(dims, indexmap, xstart_i, ystart_i, zstart_i) private(i, ii, ii2, ij2, j, jj, k, kk) schedule(static)
#pragma cuda gpurun noc2gmemtr(dims, indexmap) 
#pragma cuda gpurun nocudamalloc(dims, indexmap) 
#pragma cuda gpurun nocudafree(dims, indexmap) 
#pragma cuda gpurun nog2cmemtr(dims, indexmap, xstart_i, ystart_i, zstart_i) 
#pragma cuda ainfo kernelid(0) procname(compute_indexmap_clnd1) 
#pragma cuda gpurun cudafree(xstart_i, ystart_i, zstart_i) 
	compute_indexmap_clnd1_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__dims, pitch__dims, ((int (*)[256][512])gpu__indexmap__main), gpu__xstart_i, gpu__ystart_i, gpu__zstart_i);
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__zstart_i));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__ystart_i));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__xstart_i));
	/*
	   --------------------------------------------------------------------
	   c compute array of exponentials for time evolution. 
	   c-------------------------------------------------------------------
	 */
	/* #pragma omp single */
	{
		ap=(((( - 4.0)*1.0E-6)*3.141592653589793)*3.141592653589793);
		ex[0]=1.0;
		ex[1]=exp(ap);
#pragma loop name compute_indexmap#1 
		for (i=2; i<=(20*((((512*512)/4)+((256*256)/4))+((256*256)/4))); i ++ )
		{
			ex[i]=(ex[(i-1)]*ex[1]);
		}
	}
	/* end single */
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
static void print_timers(void )
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int i;
	char * tstrings[] =  { "          total ", "          setup ", "            fft ", "         evolve ", "       checksum ", "         fftlow ", "        fftcopy " } ;
#pragma loop name print_timers#0 
	for (i=0; i<7; i ++ )
	{
		if ((timer_read(i)!=0.0))
		{
			printf("timer %2d(%16s( :%10.6f\n", i, tstrings[i], timer_read(i));
		}
	}
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
static void fft(int dir, double x1_real[256][256][512], double x1_imag[256][256][512], double x2_real[256][256][512], double x2_imag[256][256][512])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/* dcomplex y0[NX][FFTBLOCKPAD]; */
	/* dcomplex y0[NX][FFTBLOCKPAD]; */
	/* dcomplex y1[NX][FFTBLOCKPAD]; */
	/*
	   --------------------------------------------------------------------
	   c note: args x1, x2 must be different arrays
	   c note: args for cfftsx are (direction, layout, xin, xout, scratch)
	   c       xinxout may be the same and it can be somewhat faster
	   c       if they are
	   c-------------------------------------------------------------------
	 */
	if ((dir==1))
	{
		/* cffts1(1, dims[0], x1, x1, y0, y1);	x1 -> x1 */
		cffts1(1, dims[0], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts2(1, dims[1], x1, x1, y0, y1);	x1 -> x1 */
		cffts2(1, dims[1], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts3(1, dims[2], x1, x2, y0, y1);	x1 -> x2 */
		cffts3(1, dims[2], x1_real, x1_imag, x2_real, x2_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
	}
	else
	{
		/* cffts3(-1, dims[2], x1, x1, y0, y1);	x1 -> x1 */
		cffts3_clnd1(( - 1), dims[2], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts2(-1, dims[1], x1, x1, y0, y1);	x1 -> x1 */
		cffts2_clnd1(( - 1), dims[1], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts1(-1, dims[0], x1, x2, y0, y1);	x1 -> x2 */
		cffts1_clnd1(( - 1), dims[0], x1_real, x1_imag, x2_real, x2_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x2 */
	}
	return ;
}

static void fft_clnd2_cloned0(int dir, double x1_real[256][256][512], double x1_imag[256][256][512], double x2_real[256][256][512], double x2_imag[256][256][512])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/* dcomplex y0[NX][FFTBLOCKPAD]; */
	/* dcomplex y0[NX][FFTBLOCKPAD]; */
	/* dcomplex y1[NX][FFTBLOCKPAD]; */
	/*
	   --------------------------------------------------------------------
	   c note: args x1, x2 must be different arrays
	   c note: args for cfftsx are (direction, layout, xin, xout, scratch)
	   c       xinxout may be the same and it can be somewhat faster
	   c       if they are
	   c-------------------------------------------------------------------
	 */
	if ((dir==1))
	{
		/* cffts1(1, dims[0], x1, x1, y0, y1);	x1 -> x1 */
		cffts1_clnd2_cloned0(1, dims[0], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts2(1, dims[1], x1, x1, y0, y1);	x1 -> x1 */
		cffts2_clnd2_cloned0(1, dims[1], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts3(1, dims[2], x1, x2, y0, y1);	x1 -> x2 */
		cffts3_clnd2_cloned0(1, dims[2], x1_real, x1_imag, x2_real, x2_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
	}
	else
	{
		/* cffts3(-1, dims[2], x1, x1, y0, y1);	x1 -> x1 */
		cffts3_clnd3_cloned0(( - 1), dims[2], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts2(-1, dims[1], x1, x1, y0, y1);	x1 -> x1 */
		cffts2_clnd3_cloned0(( - 1), dims[1], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts1(-1, dims[0], x1, x2, y0, y1);	x1 -> x2 */
		cffts1_clnd3_cloned0(( - 1), dims[0], x1_real, x1_imag, x2_real, x2_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x2 */
	}
	return ;
}

static void fft_clnd1(int dir, double x1_real[256][256][512], double x1_imag[256][256][512], double x2_real[256][256][512], double x2_imag[256][256][512])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/* dcomplex y0[NX][FFTBLOCKPAD]; */
	/* dcomplex y0[NX][FFTBLOCKPAD]; */
	/* dcomplex y1[NX][FFTBLOCKPAD]; */
	/*
	   --------------------------------------------------------------------
	   c note: args x1, x2 must be different arrays
	   c note: args for cfftsx are (direction, layout, xin, xout, scratch)
	   c       xinxout may be the same and it can be somewhat faster
	   c       if they are
	   c-------------------------------------------------------------------
	 */
	if ((dir==1))
	{
		/* cffts1(1, dims[0], x1, x1, y0, y1);	x1 -> x1 */
		cffts1_clnd4(1, dims[0], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts2(1, dims[1], x1, x1, y0, y1);	x1 -> x1 */
		cffts2_clnd4(1, dims[1], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts3(1, dims[2], x1, x2, y0, y1);	x1 -> x2 */
		cffts3_clnd4(1, dims[2], x1_real, x1_imag, x2_real, x2_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
	}
	else
	{
		/* cffts3(-1, dims[2], x1, x1, y0, y1);	x1 -> x1 */
		cffts3_clnd5(( - 1), dims[2], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts2(-1, dims[1], x1, x1, y0, y1);	x1 -> x1 */
		cffts2_clnd5(( - 1), dims[1], x1_real, x1_imag, x1_real, x1_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x1 */
		/* cffts1(-1, dims[0], x1, x2, y0, y1);	x1 -> x2 */
		cffts1_clnd5(( - 1), dims[0], x1_real, x1_imag, x2_real, x2_imag, NULL, NULL, NULL, NULL);
		/* x1 -> x2 */
	}
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
__global__ void cffts1_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_0, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int j;
	int jj;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[0];
#pragma loop name cffts1#1#0 
		for (jj=0; jj<=(d[1]-fftblock_0); jj+=fftblock_0)
		{
#pragma loop name cffts1#1#0#0 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#0#0 
				for (i=0; i<d_0; i ++ )
				{
					yy0_real[_gtid][i][j]=x_real[k][(j+jj)][i];
					yy0_imag[_gtid][i][j]=x_imag[k][(j+jj)][i];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_0), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts1#1#0#1 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#1#0 
				for (i=0; i<d_0; i ++ )
				{
					xout_real[k][(j+jj)][i]=yy0_real[_gtid][i][j];
					xout_imag[k][(j+jj)][i]=yy0_imag[_gtid][i][j];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts1(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_0;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_0;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts1#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_0=logd[0];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMemcpy(gpu__fftblock, ( & fftblock), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMemcpy(gpu__fftblockpad, ( & fftblockpad), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_0)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_0, ( & logd_0), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=(512*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(gpu__u_imag, u_imag, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=(512*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(gpu__u_real, u_real, gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, j, jj, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts1) 
#pragma cuda gpurun registerRO(d[0], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_0) 
	cffts1_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_0, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_0));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts1_clnd5_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_0, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int j;
	int jj;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[0];
#pragma loop name cffts1#1#0 
		for (jj=0; jj<=(d[1]-fftblock_0); jj+=fftblock_0)
		{
#pragma loop name cffts1#1#0#0 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#0#0 
				for (i=0; i<d_0; i ++ )
				{
					yy0_real[_gtid][i][j]=x_real[k][(j+jj)][i];
					yy0_imag[_gtid][i][j]=x_imag[k][(j+jj)][i];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_0), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts1#1#0#1 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#1#0 
				for (i=0; i<d_0; i ++ )
				{
					xout_real[k][(j+jj)][i]=yy0_real[_gtid][i][j];
					xout_imag[k][(j+jj)][i]=yy0_imag[_gtid][i][j];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts1_clnd5(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_0;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_0;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts1#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_0=logd[0];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_0)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_0, ( & logd_0), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, j, jj, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts1_clnd5) 
#pragma cuda gpurun registerRO(d[0], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_0) 
	cffts1_clnd5_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_0, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u0_imag__main), ((double (*)[256][512])gpu__u0_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_0));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts1_clnd4_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_0, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int j;
	int jj;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[0];
#pragma loop name cffts1#1#0 
		for (jj=0; jj<=(d[1]-fftblock_0); jj+=fftblock_0)
		{
#pragma loop name cffts1#1#0#0 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#0#0 
				for (i=0; i<d_0; i ++ )
				{
					yy0_real[_gtid][i][j]=x_real[k][(j+jj)][i];
					yy0_imag[_gtid][i][j]=x_imag[k][(j+jj)][i];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_0), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts1#1#0#1 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#1#0 
				for (i=0; i<d_0; i ++ )
				{
					xout_real[k][(j+jj)][i]=yy0_real[_gtid][i][j];
					xout_imag[k][(j+jj)][i]=yy0_imag[_gtid][i][j];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts1_clnd4(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_0;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_0;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts1#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_0=logd[0];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_0)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_0, ( & logd_0), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=(512*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(gpu__u_imag, u_imag, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=(512*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(gpu__u_real, u_real, gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, j, jj, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts1_clnd4) 
#pragma cuda gpurun registerRO(d[0], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_0) 
	cffts1_clnd4_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_0, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_0));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts1_clnd3_cloned0_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_0, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int j;
	int jj;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[0];
#pragma loop name cffts1#1#0 
		for (jj=0; jj<=(d[1]-fftblock_0); jj+=fftblock_0)
		{
#pragma loop name cffts1#1#0#0 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#0#0 
				for (i=0; i<d_0; i ++ )
				{
					yy0_real[_gtid][i][j]=x_real[k][(j+jj)][i];
					yy0_imag[_gtid][i][j]=x_imag[k][(j+jj)][i];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_0), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts1#1#0#1 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#1#0 
				for (i=0; i<d_0; i ++ )
				{
					xout_real[k][(j+jj)][i]=yy0_real[_gtid][i][j];
					xout_imag[k][(j+jj)][i]=yy0_imag[_gtid][i][j];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts1_clnd3_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_0;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_0;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts1#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_0=logd[0];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_0)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_0, ( & logd_0), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, j, jj, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun multisrccg(xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts1_clnd3_cloned0) 
#pragma cuda gpurun registerRO(d[0], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_0) 
	cffts1_clnd3_cloned0_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_0, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u2_imag__main), ((double (*)[256][512])gpu__u2_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_0));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts1_clnd2_cloned0_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_0, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int j;
	int jj;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[0];
#pragma loop name cffts1#1#0 
		for (jj=0; jj<=(d[1]-fftblock_0); jj+=fftblock_0)
		{
#pragma loop name cffts1#1#0#0 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#0#0 
				for (i=0; i<d_0; i ++ )
				{
					yy0_real[_gtid][i][j]=x_real[k][(j+jj)][i];
					yy0_imag[_gtid][i][j]=x_imag[k][(j+jj)][i];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_0), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts1#1#0#1 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#1#0 
				for (i=0; i<d_0; i ++ )
				{
					xout_real[k][(j+jj)][i]=yy0_real[_gtid][i][j];
					xout_imag[k][(j+jj)][i]=yy0_imag[_gtid][i][j];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts1_clnd2_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_0;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_0;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts1#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_0=logd[0];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_0)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_0, ( & logd_0), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, j, jj, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts1_clnd2_cloned0) 
#pragma cuda gpurun registerRO(d[0], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_0) 
	cffts1_clnd2_cloned0_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_0, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_0));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts1_clnd1_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_0, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int j;
	int jj;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[0];
#pragma loop name cffts1#1#0 
		for (jj=0; jj<=(d[1]-fftblock_0); jj+=fftblock_0)
		{
#pragma loop name cffts1#1#0#0 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#0#0 
				for (i=0; i<d_0; i ++ )
				{
					yy0_real[_gtid][i][j]=x_real[k][(j+jj)][i];
					yy0_imag[_gtid][i][j]=x_imag[k][(j+jj)][i];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_0), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts1#1#0#1 
			for (j=0; j<fftblock_0; j ++ )
			{
#pragma loop name cffts1#1#0#1#0 
				for (i=0; i<d_0; i ++ )
				{
					xout_real[k][(j+jj)][i]=yy0_real[_gtid][i][j];
					xout_imag[k][(j+jj)][i]=yy0_imag[_gtid][i][j];
				}
			}
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts1_clnd1(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_0;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_0;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts1#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_0=logd[0];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_0)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_0, ( & logd_0), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, j, jj, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_0, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts1_clnd1) 
#pragma cuda gpurun registerRO(d[0], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_0) 
	cffts1_clnd1_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_0, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u0_imag__main), ((double (*)[256][512])gpu__u0_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_0));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
__global__ void cffts2_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_1, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[1];
#pragma loop name cffts2#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#0 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][j][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][j][i]=x_imag[k][j][(i+ii)];
				}
			}
			/* 	    if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_1), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#1 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][j][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][j][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts2(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_1;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_1;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts2#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_1=logd[1];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_1)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_1, ( & logd_1), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts2) 
#pragma cuda gpurun registerRO(d[1], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_1) 
	cffts2_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_1, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_1));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts2_clnd5_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_1, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[1];
#pragma loop name cffts2#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#0 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][j][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][j][i]=x_imag[k][j][(i+ii)];
				}
			}
			/* 	    if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_1), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#1 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][j][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][j][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts2_clnd5(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_1;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_1;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts2#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_1=logd[1];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_1)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_1, ( & logd_1), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts2_clnd5) 
#pragma cuda gpurun registerRO(d[1], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_1) 
	cffts2_clnd5_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_1, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_1));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts2_clnd4_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_1, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[1];
#pragma loop name cffts2#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#0 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][j][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][j][i]=x_imag[k][j][(i+ii)];
				}
			}
			/* 	    if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_1), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#1 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][j][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][j][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts2_clnd4(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_1;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_1;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts2#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_1=logd[1];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_1)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_1, ( & logd_1), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts2_clnd4) 
#pragma cuda gpurun registerRO(d[1], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_1) 
	cffts2_clnd4_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_1, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_1));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts2_clnd3_cloned0_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_1, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[1];
#pragma loop name cffts2#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#0 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][j][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][j][i]=x_imag[k][j][(i+ii)];
				}
			}
			/* 	    if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_1), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#1 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][j][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][j][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts2_clnd3_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_1;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_1;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts2#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_1=logd[1];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_1)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_1, ( & logd_1), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts2_clnd3_cloned0) 
#pragma cuda gpurun registerRO(d[1], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_1) 
	cffts2_clnd3_cloned0_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_1, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_1));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts2_clnd2_cloned0_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_1, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[1];
#pragma loop name cffts2#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#0 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][j][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][j][i]=x_imag[k][j][(i+ii)];
				}
			}
			/* 	    if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_1), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#1 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][j][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][j][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts2_clnd2_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_1;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_1;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts2#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_1=logd[1];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_1)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_1, ( & logd_1), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts2_clnd2_cloned0) 
#pragma cuda gpurun registerRO(d[1], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_1) 
	cffts2_clnd2_cloned0_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_1, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_1));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts2_clnd1_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_1, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	k=_gtid;
	fftblock_0=( * fftblock);
	if (k<d[2])
	{
		d_0=d[1];
#pragma loop name cffts2#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#0 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][j][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][j][i]=x_imag[k][j][(i+ii)];
				}
			}
			/* 	    if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_1), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*          if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*          if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts2#1#0#1 
			for (j=0; j<d_0; j ++ )
			{
#pragma loop name cffts2#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][j][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][j][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts2_clnd1(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_1;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_1;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts2#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_1=logd[1];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_1)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_1, ( & logd_1), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[2])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_1, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts2_clnd1) 
#pragma cuda gpurun registerRO(d[1], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_1) 
	cffts2_clnd1_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_1, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_1));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
__global__ void cffts3_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_2, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	j=_gtid;
	fftblock_0=( * fftblock);
	if (j<d[1])
	{
		d_0=d[2];
#pragma loop name cffts3#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#0 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][k][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][k][i]=x_imag[k][j][(i+ii)];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_2), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#1 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][k][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][k][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts3(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_2;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_2;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts3#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_2=logd[2];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_2)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_2, ( & logd_2), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[1])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts3) 
#pragma cuda gpurun registerRO(d[2], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_2) 
	cffts3_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_2, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u0_imag__main), ((double (*)[256][512])gpu__u0_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_2));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts3_clnd5_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_2, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	j=_gtid;
	fftblock_0=( * fftblock);
	if (j<d[1])
	{
		d_0=d[2];
#pragma loop name cffts3#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#0 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][k][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][k][i]=x_imag[k][j][(i+ii)];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_2), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#1 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][k][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][k][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts3_clnd5(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_2;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_2;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts3#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_2=logd[2];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_2)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_2, ( & logd_2), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=(512*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(gpu__u_imag, u_imag, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=(512*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(gpu__u_real, u_real, gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[1])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts3_clnd5) 
#pragma cuda gpurun registerRO(d[2], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_2) 
	cffts3_clnd5_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_2, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_2));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts3_clnd4_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_2, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	j=_gtid;
	fftblock_0=( * fftblock);
	if (j<d[1])
	{
		d_0=d[2];
#pragma loop name cffts3#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#0 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][k][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][k][i]=x_imag[k][j][(i+ii)];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_2), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#1 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][k][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][k][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts3_clnd4(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_2;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_2;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts3#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_2=logd[2];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_2)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_2, ( & logd_2), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[1])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts3_clnd4) 
#pragma cuda gpurun registerRO(d[2], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_2) 
	cffts3_clnd4_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_2, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u0_imag__main), ((double (*)[256][512])gpu__u0_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_2));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts3_clnd3_cloned0_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_2, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	j=_gtid;
	fftblock_0=( * fftblock);
	if (j<d[1])
	{
		d_0=d[2];
#pragma loop name cffts3#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#0 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][k][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][k][i]=x_imag[k][j][(i+ii)];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_2), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#1 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][k][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][k][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts3_clnd3_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_2;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_2;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts3#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_2=logd[2];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_2)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_2, ( & logd_2), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[1])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts3_clnd3_cloned0) 
#pragma cuda gpurun registerRO(d[2], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_2) 
	cffts3_clnd3_cloned0_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_2, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_2));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts3_clnd2_cloned0_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_2, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	j=_gtid;
	fftblock_0=( * fftblock);
	if (j<d[1])
	{
		d_0=d[2];
#pragma loop name cffts3#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#0 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][k][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][k][i]=x_imag[k][j][(i+ii)];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_2), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#1 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][k][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][k][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts3_clnd2_cloned0(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_2;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_2;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts3#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_2=logd[2];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_2)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_2, ( & logd_2), gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[1])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun multisrccg(xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts3_clnd2_cloned0) 
#pragma cuda gpurun registerRO(d[2], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_2) 
	cffts3_clnd2_cloned0_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_2, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u2_imag__main), ((double (*)[256][512])gpu__u2_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_2));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

__global__ void cffts3_clnd1_kernel0(int * d, int * fftblock, int * fftblockpad, int * is, int * logd_2, double * u_imag, double * u_real, double x_imag[256][256][512], double x_real[256][256][512], double xout_imag[256][256][512], double xout_real[256][256][512], double yy0_imag[][512][18], double yy0_real[][512][18], double yy1_imag[][512][18], double yy1_real[][512][18])
{
	int d_0;
	int fftblock_0;
	int i;
	int ii;
	int j;
	int k;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	j=_gtid;
	fftblock_0=( * fftblock);
	if (j<d[1])
	{
		d_0=d[2];
#pragma loop name cffts3#1#0 
		for (ii=0; ii<=(d[0]-fftblock_0); ii+=fftblock_0)
		{
			/* 	    if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#0 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#0#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					yy0_real[_gtid][k][i]=x_real[k][j][(i+ii)];
					yy0_imag[_gtid][k][i]=x_imag[k][j][(i+ii)];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTLOW); */
			dev_cfftz(( * is), ( * logd_2), d_0, yy0_real, yy0_imag, yy1_real, yy1_imag, fftblock, fftblockpad, u_imag, u_real, _gtid);
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTLOW); */
			/*           if (TIMERS_ENABLED == TRUE) timer_start(T_FFTCOPY); */
#pragma loop name cffts3#1#0#1 
			for (k=0; k<d_0; k ++ )
			{
#pragma loop name cffts3#1#0#1#0 
				for (i=0; i<fftblock_0; i ++ )
				{
					xout_real[k][j][(i+ii)]=yy0_real[_gtid][k][i];
					xout_imag[k][j][(i+ii)]=yy0_imag[_gtid][k][i];
				}
			}
			/*           if (TIMERS_ENABLED == TRUE) timer_stop(T_FFTCOPY); */
		}
	}
}

static void cffts3_clnd1(int is, int d[3], double x_real[256][256][512], double x_imag[256][256][512], double xout_real[256][256][512], double xout_imag[256][256][512], double y0_real[512][18], double y0_imag[512][18], double y1_real[512][18], double y1_imag[512][18])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int logd[3];
	int i;
	int logd_2;
	int * gpu__d;
	int * gpu__is;
	int * gpu__logd_2;
	double * gpu__yy0_imag;
	double * gpu__yy0_real;
	double * gpu__yy1_imag;
	double * gpu__yy1_real;
#pragma loop name cffts3#0 
	for (i=0; i<3; i ++ )
	{
		logd[i]=ilog2(d[i]);
	}
	logd_2=logd[2];
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__d)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__d, d, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMemcpy(gpu__fftblock, ( & fftblock), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMemcpy(gpu__fftblockpad, ( & fftblockpad), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__is)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__is, ( & is), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__logd_2)), gpuBytes));
	CUDA_SAFE_CALL(hipMemcpy(gpu__logd_2, ( & logd_2), gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=(512*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(gpu__u_imag, u_imag, gpuBytes, hipMemcpyHostToDevice));
	gpuBytes=(512*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(gpu__u_real, u_real, gpuBytes, hipMemcpyHostToDevice));
	dim3 dimBlock0(gpuNumThreads, 1, 1);
	gpuNumBlocks=((int)ceil((((float)d[1])/1024.0F)));
	if ((gpuNumBlocks>MAX_GDIMENSION))
	{
		gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
		gpuNumBlocks1=MAX_NDIMENSION;
	}
	else
	{
		gpuNumBlocks2=1;
		gpuNumBlocks1=gpuNumBlocks;
	}
	dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
	gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
	totalNumThreads=(gpuNumBlocks*gpuNumThreads);
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy0_real)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_imag)), gpuBytes));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipMalloc(((void *  * )( & gpu__yy1_real)), gpuBytes));
#pragma omp parallel for threadprivate(yy0_imag, yy0_real, yy1_imag, yy1_real) shared(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) private(i, ii, j, k) schedule(static)
#pragma cuda gpurun noc2gmemtr(x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudafree(fftblock, fftblockpad, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda gpurun nog2cmemtr(d, fftblock, fftblockpad, is, logd_2, u_imag, u_real, x_imag, x_real, xout_imag, xout_real) 
#pragma cuda ainfo kernelid(0) procname(cffts3_clnd1) 
#pragma cuda gpurun registerRO(d[2], fftblock) 
#pragma cuda gpurun cudafree(d, is, logd_2) 
	cffts3_clnd1_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__d, gpu__fftblock, gpu__fftblockpad, gpu__is, gpu__logd_2, gpu__u_imag, gpu__u_real, ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[256][512])gpu__u1_imag__main), ((double (*)[256][512])gpu__u1_real__main), ((double (*)[512][18])gpu__yy0_imag), ((double (*)[512][18])gpu__yy0_real), ((double (*)[512][18])gpu__yy1_imag), ((double (*)[512][18])gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_real, gpu__yy1_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy1_imag, gpu__yy1_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy1_imag));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_real, gpu__yy0_real, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_real));
	gpuBytes=((512*18)*sizeof (double));
	CUDA_SAFE_CALL(hipMemcpy(yy0_imag, gpu__yy0_imag, gpuBytes, hipMemcpyDeviceToHost));
	gpuBytes=(totalNumThreads*((512*18)*sizeof (double)));
	CUDA_SAFE_CALL(hipFree(gpu__yy0_imag));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__logd_2));
	gpuBytes=sizeof (int);
	CUDA_SAFE_CALL(hipFree(gpu__is));
	gpuBytes=(3*sizeof (int));
	CUDA_SAFE_CALL(hipFree(gpu__d));
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
static void fft_init(int n)
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/*
	   --------------------------------------------------------------------
	   c compute the roots-of-unity array that will be used for subsequent FFTs. 
	   c-------------------------------------------------------------------
	 */
	int m;
	int nu;
	int ku;
	int i;
	int j;
	int ln;
	double t;
	double ti;
	/*
	   --------------------------------------------------------------------
	   c   Initialize the U array with sines and cosines in a manner that permits
	   c   stride one access at each FFT iteration.
	   c-------------------------------------------------------------------
	 */
	nu=n;
	m=ilog2(n);
	u_real[0]=((double)m);
	u_imag[0]=0.0;
	ku=1;
	ln=1;
#pragma loop name fft_init#0 
	for (j=1; j<=m; j ++ )
	{
		t=(3.141592653589793/ln);
#pragma loop name fft_init#0#0 
		for (i=0; i<=(ln-1); i ++ )
		{
			ti=(i*t);
			u_real[(i+ku)]=cos(ti);
			u_imag[(i+ku)]=sin(ti);
		}
		ku=(ku+ln);
		ln=(2*ln);
	}
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
__device__ static void dev_cfftz(int is, int m, int n, double x_real[][512][18], double x_imag[][512][18], double y_real[][512][18], double y_imag[][512][18], int * fftblock, int * fftblockpad, double u_imag[512], double u_real[512], int _gtid)
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/*
	   --------------------------------------------------------------------
	   c   Computes NY N-point complex-to-complex FFTs of X using an algorithm due
	   c   to Swarztrauber.  X is both the input and the output array, while Y is a 
	   c   scratch array.  It is assumed that N = 2^M.  Before calling CFFTZ to 
	   c   perform FFTs, the array U must be initialized by calling CFFTZ with IS 
	   c   set to 0 and M set to MX, where MX is the maximum value of M for any 
	   c   subsequent call.
	   c-------------------------------------------------------------------
	 */
	int i;
	int j;
	int l;
	int mx;
	/*
	   --------------------------------------------------------------------
	   c   Check if input parameters are invalid.
	   c-------------------------------------------------------------------
	 */
	int fftblock_0;
	fftblock_0=( * fftblock);
	mx=((int)u_real[0]);
	/*
	   --------------------------------------------------------------------
	   c   Perform one variant of the Stockham FFT.
	   c-------------------------------------------------------------------
	 */
#pragma loop name cfftz#0 
	for (l=1; l<=m; l+=2)
	{
		dev_fftz2(is, l, m, n, fftblock_0, ( * fftblockpad), u_real, u_imag, x_real, x_imag, y_real, y_imag, _gtid);
		if ((l==m))
		{
			break;
		}
		dev_fftz2(is, (l+1), m, n, fftblock_0, ( * fftblockpad), u_real, u_imag, y_real, y_imag, x_real, x_imag, _gtid);
	}
	/*
	   --------------------------------------------------------------------
	   c   Copy Y to X.
	   c-------------------------------------------------------------------
	 */
	if (((m%2)==1))
	{
#pragma loop name cfftz#1 
		for (j=0; j<n; j ++ )
		{
#pragma loop name cfftz#1#0 
			for (i=0; i<fftblock_0; i ++ )
			{
				x_real[_gtid][j][i]=y_real[_gtid][j][i];
				x_imag[_gtid][j][i]=y_imag[_gtid][j][i];
			}
		}
	}
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
__device__ static void dev_fftz2(int is, int l, int m, int n, int ny, int ny1, double u_real[512], double u_imag[512], double x_real[][512][18], double x_imag[][512][18], double y_real[][512][18], double y_imag[][512][18], int _gtid)
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	/*
	   --------------------------------------------------------------------
	   c   Performs the L-th iteration of the second variant of the Stockham FFT.
	   c-------------------------------------------------------------------
	 */
	int k;
	int n1;
	int li;
	int lj;
	int lk;
	int ku;
	int i;
	int j;
	int i11;
	int i12;
	int i21;
	int i22;
	double u1_real;
	double u1_imag;
	/*
	   --------------------------------------------------------------------
	   c   Set initial parameters.
	   c-------------------------------------------------------------------
	 */
	n1=(n/2);
	if (((l-1)==0))
	{
		lk=1;
	}
	else
	{
		lk=(2<<((l-1)-1));
	}
	if (((m-l)==0))
	{
		li=1;
	}
	else
	{
		li=(2<<((m-l)-1));
	}
	lj=(2*lk);
	ku=li;
#pragma loop name fftz2#0 
	for (i=0; i<li; i ++ )
	{
		i11=(i*lk);
		i12=(i11+n1);
		i21=(i*lj);
		i22=(i21+lk);
		if ((is>=1))
		{
			u1_real=u_real[(ku+i)];
			u1_imag=u_imag[(ku+i)];
		}
		else
		{
			u1_real=u_real[(ku+i)];
			u1_imag=( - u_imag[(ku+i)]);
		}
		/*
		   --------------------------------------------------------------------
		   c   This loop is vectorizable.
		   c-------------------------------------------------------------------
		 */
#pragma loop name fftz2#0#0 
		for (k=0; k<lk; k ++ )
		{
#pragma loop name fftz2#0#0#0 
			for (j=0; j<ny; j ++ )
			{
				double x11real;
				double x11imag;
				double x21real;
				double x21imag;
				x11real=x_real[_gtid][(i11+k)][j];
				x11imag=x_imag[_gtid][(i11+k)][j];
				x21real=x_real[_gtid][(i12+k)][j];
				x21imag=x_imag[_gtid][(i12+k)][j];
				y_real[_gtid][(i21+k)][j]=(x11real+x21real);
				y_imag[_gtid][(i21+k)][j]=(x11imag+x21imag);
				y_real[_gtid][(i22+k)][j]=((u1_real*(x11real-x21real))-(u1_imag*(x11imag-x21imag)));
				y_imag[_gtid][(i22+k)][j]=((u1_real*(x11imag-x21imag))+(u1_imag*(x11real-x21real)));
			}
		}
	}
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
static int ilog2(int n)
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int nn;
	int lg;
	int _ret_val_0;
	if ((n==1))
	{
		_ret_val_0=0;
		return _ret_val_0;
	}
	lg=1;
	nn=2;
	while (nn<n)
	{
		nn=(nn<<1);
		lg ++ ;
	}
	return lg;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
__global__ void checksum_kernel0(double * red__chk_imag, double * red__chk_real, double u1_imag[256][256][512], double u1_real[256][256][512], int * xend, int * xstart, int * yend, int * ystart, int * zend, int * zstart)
{
	__shared__ double sh__chk_imag[BLOCK_SIZE];
	__shared__ double sh__chk_real[BLOCK_SIZE];
	int xstart_0;
	int ystart_0;
	int zstart_0;
	int j;
	int q;
	int r;
	int s;
	int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
	int _gtid = (threadIdx.x+(_bid*blockDim.x));
	zstart_0=zstart[0];
	ystart_0=ystart[0];
	xstart_0=xstart[0];
	sh__chk_real[threadIdx.x]=0.0F;
	sh__chk_imag[threadIdx.x]=0.0F;
	j=(_gtid+1);
#pragma omp for nowait
	if (j<=1024)
	{
		q=((j%512)+1);
		if (((q>=xstart_0)&&(q<=xend[0])))
		{
			r=(((3*j)%256)+1);
			if (((r>=ystart_0)&&(r<=yend[0])))
			{
				s=(((5*j)%256)+1);
				if (((s>=zstart_0)&&(s<=zend[0])))
				{
					/* cadd is a macro in npb-C.h adding the real and imaginary */
					/* component. So, the preprocessed statement still follows the */
					/* reduction pattern */
					/* cadd(chk,chk,u1[s-zstart[0]][r-ystart[0]][q-xstart[0]]); */
					sh__chk_real[threadIdx.x]=(sh__chk_real[threadIdx.x]+u1_real[(s-zstart_0)][(r-ystart_0)][(q-xstart_0)]);
					sh__chk_imag[threadIdx.x]=(sh__chk_imag[threadIdx.x]+u1_imag[(s-zstart_0)][(r-ystart_0)][(q-xstart_0)]);
				}
			}
		}
	}
	__syncthreads();
	if ((threadIdx.x<256))
	{
		sh__chk_imag[threadIdx.x]+=sh__chk_imag[(threadIdx.x+256)];
		sh__chk_real[threadIdx.x]+=sh__chk_real[(threadIdx.x+256)];
	}
	__syncthreads();
	if ((threadIdx.x<128))
	{
		sh__chk_imag[threadIdx.x]+=sh__chk_imag[(threadIdx.x+128)];
		sh__chk_real[threadIdx.x]+=sh__chk_real[(threadIdx.x+128)];
	}
	__syncthreads();
	if ((threadIdx.x<64))
	{
		sh__chk_imag[threadIdx.x]+=sh__chk_imag[(threadIdx.x+64)];
		sh__chk_real[threadIdx.x]+=sh__chk_real[(threadIdx.x+64)];
	}
	__syncthreads();
	if ((threadIdx.x<32))
	{
		sh__chk_imag[threadIdx.x]+=sh__chk_imag[(threadIdx.x+32)];
		sh__chk_real[threadIdx.x]+=sh__chk_real[(threadIdx.x+32)];
	}
	if ((threadIdx.x<16))
	{
		sh__chk_imag[threadIdx.x]+=sh__chk_imag[(threadIdx.x+16)];
		sh__chk_real[threadIdx.x]+=sh__chk_real[(threadIdx.x+16)];
	}
	if ((threadIdx.x<8))
	{
		sh__chk_imag[threadIdx.x]+=sh__chk_imag[(threadIdx.x+8)];
		sh__chk_real[threadIdx.x]+=sh__chk_real[(threadIdx.x+8)];
	}
	if ((threadIdx.x<4))
	{
		sh__chk_imag[threadIdx.x]+=sh__chk_imag[(threadIdx.x+4)];
		sh__chk_real[threadIdx.x]+=sh__chk_real[(threadIdx.x+4)];
	}
	if ((threadIdx.x<2))
	{
		sh__chk_imag[threadIdx.x]+=sh__chk_imag[(threadIdx.x+2)];
		sh__chk_real[threadIdx.x]+=sh__chk_real[(threadIdx.x+2)];
	}
	if ((threadIdx.x<1))
	{
		sh__chk_imag[threadIdx.x]+=sh__chk_imag[(threadIdx.x+1)];
		sh__chk_real[threadIdx.x]+=sh__chk_real[(threadIdx.x+1)];
	}
	if ((threadIdx.x==0))
	{
		red__chk_imag[_bid]=sh__chk_imag[0];
		red__chk_real[_bid]=sh__chk_real[0];
	}
}

static void checksum(int i, double u1_real[256][256][512], double u1_imag[256][256][512], int d[3])
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	double _chk_real;
	double _chk_imag;
	double * red__chk_imag;
	double * chk_imag__extended;
	int _ti_100_0;
	double * red__chk_real;
	double * chk_real__extended;
	_chk_real=0.0;
	_chk_imag=0.0;
	{
		double chk_real = _chk_real;
		double chk_imag = _chk_imag;
		/* #pragma omp for nowait */
		dim3 dimBlock0(gpuNumThreads, 1, 1);
		gpuNumBlocks=1;
		if ((gpuNumBlocks>MAX_GDIMENSION))
		{
			gpuNumBlocks2=1;
			gpuNumBlocks1=MAX_NDIMENSION;
		}
		else
		{
			gpuNumBlocks2=1;
			gpuNumBlocks1=gpuNumBlocks;
		}
		dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
		gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
		totalNumThreads=(gpuNumBlocks*gpuNumThreads);
		gpuBytes=(gpuNumBlocks*sizeof (double));
		CUDA_SAFE_CALL(hipMalloc(((void *  * )( & red__chk_imag)), gpuBytes));
		chk_imag__extended=((double * )malloc(gpuBytes));
		gpuBytes=(gpuNumBlocks*sizeof (double));
		CUDA_SAFE_CALL(hipMalloc(((void *  * )( & red__chk_real)), gpuBytes));
		chk_real__extended=((double * )malloc(gpuBytes));
		gpuBytes=(3*sizeof (int));
		CUDA_SAFE_CALL(hipMemcpy(gpu__xend, xend, gpuBytes, hipMemcpyHostToDevice));
		gpuBytes=(3*sizeof (int));
		CUDA_SAFE_CALL(hipMemcpy(gpu__xstart, xstart, gpuBytes, hipMemcpyHostToDevice));
		gpuBytes=(3*sizeof (int));
		CUDA_SAFE_CALL(hipMemcpy(gpu__yend, yend, gpuBytes, hipMemcpyHostToDevice));
		gpuBytes=(3*sizeof (int));
		CUDA_SAFE_CALL(hipMemcpy(gpu__ystart, ystart, gpuBytes, hipMemcpyHostToDevice));
		gpuBytes=(3*sizeof (int));
		CUDA_SAFE_CALL(hipMemcpy(gpu__zend, zend, gpuBytes, hipMemcpyHostToDevice));
		gpuBytes=(3*sizeof (int));
		CUDA_SAFE_CALL(hipMemcpy(gpu__zstart, zstart, gpuBytes, hipMemcpyHostToDevice));
#pragma omp parallel shared(u1_imag, u1_real, xend, xstart, yend, ystart, zend, zstart) private(j, q, r, s) reduction(+: chk_imag, chk_real) schedule(static)
#pragma cuda gpurun noc2gmemtr(u1_imag, u1_real, x_imag, x_real, xout_imag, xout_real, yy0_imag, yy0_real) 
#pragma cuda gpurun nocudamalloc(u1_imag, u1_real) 
#pragma cuda gpurun nocudafree(u1_imag, u1_real, xend, xstart, yend, ystart, zend, zstart) 
#pragma cuda gpurun multisrccg(xend, xstart, yend, ystart, zend, zstart) 
#pragma cuda gpurun nog2cmemtr(u1_imag, u1_real, xend, xstart, yend, ystart, zend, zstart) 
#pragma cuda ainfo kernelid(0) procname(checksum) 
#pragma cuda gpurun registerRO(xstart[0], ystart[0], zstart[0]) 
		checksum_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(red__chk_imag, red__chk_real, ((double (*)[256][512])gpu__u2_imag__main), ((double (*)[256][512])gpu__u2_real__main), gpu__xend, gpu__xstart, gpu__yend, gpu__ystart, gpu__zend, gpu__zstart);
		gpuBytes=(gpuNumBlocks*sizeof (double));
		CUDA_SAFE_CALL(hipMemcpy(chk_real__extended, red__chk_real, gpuBytes, hipMemcpyDeviceToHost));
		for (_ti_100_0=0; _ti_100_0<gpuNumBlocks; _ti_100_0 ++ )
		{
			chk_real+=chk_real__extended[_ti_100_0];
		}
		free(chk_real__extended);
		CUDA_SAFE_CALL(hipFree(red__chk_real));
		gpuBytes=(gpuNumBlocks*sizeof (double));
		CUDA_SAFE_CALL(hipMemcpy(chk_imag__extended, red__chk_imag, gpuBytes, hipMemcpyDeviceToHost));
		for (_ti_100_0=0; _ti_100_0<gpuNumBlocks; _ti_100_0 ++ )
		{
			chk_imag+=chk_imag__extended[_ti_100_0];
		}
		free(chk_imag__extended);
		CUDA_SAFE_CALL(hipFree(red__chk_imag));
		_chk_real=chk_real;
		_chk_imag=chk_imag;
	}
	/* #pragma omp critical */
	{
		sums_real[i]+=_chk_real;
		sums_imag[i]+=_chk_imag;
	}
	/* #pragma omp barrier */
	/* #pragma omp single */
	{
		/* complex % real */
		sums_real[i]=(sums_real[i]/((double)33554432));
		sums_imag[i]=(sums_imag[i]/((double)33554432));
		printf("T = %5d     Checksum = %22.12e %22.12e\n", i, sums_real[i], sums_imag[i]);
	}
	return ;
}

/*
   --------------------------------------------------------------------
   c-------------------------------------------------------------------
 */
static void verify(int d1, int d2, int d3, int nt, int * verified, char * cclass)
{
	/*
	   --------------------------------------------------------------------
	   c-------------------------------------------------------------------
	 */
	int i;
	double err;
	double epsilon;
	/*
	   --------------------------------------------------------------------
	   c   Sample size reference checksums
	   c-------------------------------------------------------------------
	 */
	/*
	   --------------------------------------------------------------------
	   c   Class S size reference checksums
	   c-------------------------------------------------------------------
	 */
	double vdata_real_s[(6+1)] =  { 0.0, 554.6087004964, 554.6385409189, 554.6148406171, 554.5423607415, 554.4255039624, 554.2683411902 } ;
	double vdata_imag_s[(6+1)] =  { 0.0, 484.5363331978, 486.5304269511, 488.3910722336, 490.1273169046, 491.7475857993, 493.2597244941 } ;
	/*
	   --------------------------------------------------------------------
	   c   Class W size reference checksums
	   c-------------------------------------------------------------------
	 */
	double vdata_real_w[(6+1)] =  { 0.0, 567.3612178944, 563.1436885271, 559.402408997, 556.069804702, 553.089899125, 550.4159734538 } ;
	double vdata_imag_w[(6+1)] =  { 0.0, 529.3246849175, 528.2149986629, 527.0996558037, 526.0027904925, 524.9400845633, 523.9212247086 } ;
	/*
	   --------------------------------------------------------------------
	   c   Class A size reference checksums
	   c-------------------------------------------------------------------
	 */
	double vdata_real_a[(6+1)] =  { 0.0, 504.6735008193, 505.9412319734, 506.9376896287, 507.7892868474, 508.5233095391, 509.1487099959 } ;
	double vdata_imag_a[(6+1)] =  { 0.0, 511.404790551, 509.8809666433, 509.8144042213, 510.1336130759, 510.4914655194, 510.7917842803 } ;
	/*
	   --------------------------------------------------------------------
	   c   Class B size reference checksums
	   c-------------------------------------------------------------------
	 */
	double vdata_real_b[(20+1)] =  { 0.0, 517.7643571579, 515.4521291263, 514.6409228649, 514.2378756213, 513.9626667737, 513.7423460082, 513.5547056878, 513.3910925466, 513.247070539, 513.1197729984, 513.0070319283, 512.9070537032, 512.8182883502, 512.7393733383, 512.669106202, 512.6064276004, 512.550407657, 512.500233172, 512.4551951846, 512.4146770029 } ;
	double vdata_imag_b[(20+1)] =  { 0.0, 507.7803458597, 508.8249431599, 509.6208912659, 510.1023387619, 510.3976610617, 510.5948019802, 510.7404165783, 510.8576573661, 510.9577278523, 511.0460304483, 511.12524338, 511.1968077718, 511.2616233064, 511.3203605551, 511.3735928093, 511.4218460548, 511.465613976, 511.5053595966, 511.5415130407, 511.5744692211 } ;
	/*
	   --------------------------------------------------------------------
	   c   Class C size reference checksums
	   c-------------------------------------------------------------------
	 */
	double vdata_real_c[(20+1)] =  { 0.0, 519.5078707457, 515.5422171134, 514.4678022222, 514.0150594328, 513.755042681, 513.5811056728, 513.4569343165, 513.3651975661, 513.2955192805, 513.2410471738, 513.1971141679, 513.1605205716, 513.1290734194, 513.1012720314, 513.0760908195, 513.0528295923, 513.0310107773, 513.0103090133, 512.9905029333, 512.9714421109 } ;
	double vdata_imag_c[(20+1)] =  { 0.0, 514.9019699238, 512.7578201997, 512.2251847514, 512.1090289018, 512.1143685824, 512.1496764568, 512.1870921893, 512.2193250322, 512.2454735794, 512.2663649603, 512.2830879827, 512.2965869718, 512.3075927445, 512.3166486553, 512.3241541685, 512.3304037599, 512.3356167976, 512.3399592211, 512.3435588985, 512.3465164008 } ;
	epsilon=1.0E-12;
	( * verified)=1;
	( * cclass)='U';
	if (((((d1==64)&&(d2==64))&&(d3==64))&&(nt==6)))
	{
		( * cclass)='S';
#pragma loop name verify#0 
		for (i=1; i<=nt; i ++ )
		{
			err=((sums_real[i]-vdata_real_s[i])/vdata_real_s[i]);
			if ((fabs(err)>epsilon))
			{
				( * verified)=0;
				break;
			}
			err=((sums_imag[i]-vdata_imag_s[i])/vdata_imag_s[i]);
			if ((fabs(err)>epsilon))
			{
				( * verified)=0;
				break;
			}
		}
	}
	else
	{
		if (((((d1==128)&&(d2==128))&&(d3==32))&&(nt==6)))
		{
			( * cclass)='W';
#pragma loop name verify#1 
			for (i=1; i<=nt; i ++ )
			{
				err=((sums_real[i]-vdata_real_w[i])/vdata_real_w[i]);
				if ((fabs(err)>epsilon))
				{
					( * verified)=0;
					break;
				}
				err=((sums_imag[i]-vdata_imag_w[i])/vdata_imag_w[i]);
				if ((fabs(err)>epsilon))
				{
					( * verified)=0;
					break;
				}
			}
		}
		else
		{
			if (((((d1==256)&&(d2==256))&&(d3==128))&&(nt==6)))
			{
				( * cclass)='A';
#pragma loop name verify#2 
				for (i=1; i<=nt; i ++ )
				{
					err=((sums_real[i]-vdata_real_a[i])/vdata_real_a[i]);
					if ((fabs(err)>epsilon))
					{
						( * verified)=0;
						break;
					}
					err=((sums_imag[i]-vdata_imag_a[i])/vdata_imag_a[i]);
					if ((fabs(err)>epsilon))
					{
						( * verified)=0;
						break;
					}
				}
			}
			else
			{
				if (((((d1==512)&&(d2==256))&&(d3==256))&&(nt==20)))
				{
					( * cclass)='B';
#pragma loop name verify#3 
					for (i=1; i<=nt; i ++ )
					{
						err=((sums_real[i]-vdata_real_b[i])/vdata_real_b[i]);
						if ((fabs(err)>epsilon))
						{
							( * verified)=0;
							break;
						}
						err=((sums_imag[i]-vdata_imag_b[i])/vdata_imag_b[i]);
						if ((fabs(err)>epsilon))
						{
							( * verified)=0;
							break;
						}
					}
				}
				else
				{
					if (((((d1==512)&&(d2==512))&&(d3==512))&&(nt==20)))
					{
						( * cclass)='C';
#pragma loop name verify#4 
						for (i=1; i<=nt; i ++ )
						{
							err=((sums_real[i]-vdata_real_c[i])/vdata_real_c[i]);
							if ((fabs(err)>epsilon))
							{
								( * verified)=0;
								break;
							}
							err=((sums_imag[i]-vdata_imag_c[i])/vdata_imag_c[i]);
							if ((fabs(err)>epsilon))
							{
								( * verified)=0;
								break;
							}
						}
					}
				}
			}
		}
	}
	if ((( * cclass)!='U'))
	{
		printf("Result verification successful\n");
	}
	else
	{
		printf("Result verification failed\n");
	}
	printf("cclass = %1c\n", ( * cclass));
	return ;
}

